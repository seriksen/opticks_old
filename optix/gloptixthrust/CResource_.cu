
#include "CResource.hh"
#include "assert.h"
#include <cuda_gl_interop.h>
#include "hip/hip_runtime_api.h"

struct CResourceImp {
   unsigned int buffer_id ; 
   size_t       bufsize  ; 
   unsigned int flags ; 
   hipStream_t stream ; 
   struct hipGraphicsResource*  resource ;
   void*         dev_ptr ;   

   CResourceImp(unsigned int buffer_id, unsigned int flags, hipStream_t stream) : 
       buffer_id(buffer_id),
       bufsize(0),
       flags(flags),  
       stream(NULL),
       resource(NULL),
       dev_ptr(NULL)
   {
   }

   const char* getFlagDescription()
   {
       const char* ret(NULL);
       switch(flags)
       {
           case cudaGraphicsMapFlagsNone:         ret="cudaGraphicsMapFlagsNone: Default; Assume resource can be read/written " ; break ;
           case cudaGraphicsMapFlagsReadOnly:     ret="cudaGraphicsMapFlagsReadOnly: CUDA will not write to this resource " ; break ; 
           case cudaGraphicsMapFlagsWriteDiscard: ret="cudaGraphicsMapFlagsWriteDiscard: CUDA will only write to and will not read from this resource " ; break ;  
       }
       return ret ;
   }

   void registerBuffer()
   {
       //printf("Resource::registerBuffer %d : %s \n", buffer_id, getFlagDescription() );
       checkCudaErrors( hipGraphicsGLRegisterBuffer(&resource, buffer_id, flags) );
   }

   void unregisterBuffer()
   {
       //printf("Resource::unregisterBuffer %d \n", buffer_id );
       checkCudaErrors( hipGraphicsUnregisterResource(resource) );
   }


   void* mapGLToCUDA() 
   {
       checkCudaErrors( hipGraphicsMapResources(1, &resource, stream) );
       checkCudaErrors( hipGraphicsResourceGetMappedPointer((void **)&dev_ptr, &bufsize, resource) );
       //printf("Resource::mapGLToCUDA bufsize %lu dev_ptr %p \n", bufsize, dev_ptr );
       return dev_ptr ; 
   }

   void unmapGLToCUDA()
   {
       //printf("Resource::unmapGLToCUDA\n");
       checkCudaErrors( hipGraphicsUnmapResources(1, &resource, stream));
   } 

   void streamSync()
   {
       //printf("Resource::streamSync\n");
       checkCudaErrors( hipStreamSynchronize(stream));
   }

};


void CResource::init()
{
    unsigned int flags ;
    switch(m_access)
    {
       case RW: flags = cudaGraphicsMapFlagsNone         ;break;
       case  R: flags = cudaGraphicsMapFlagsReadOnly     ;break;
       case  W: flags = cudaGraphicsMapFlagsWriteDiscard ;break;
    }

    //hipStream_t stream1 ; 
    //hipStreamCreate ( &stream1) ;
    m_imp = new CResourceImp(m_buffer_id, flags, (hipStream_t)0  );
}


unsigned int CResource::getNumBytes()
{
    assert(m_mapped);
    return m_imp->bufsize ; 
}

void* CResource::getRawPointer()
{
    assert(m_mapped);
    return m_imp->dev_ptr ;
}

void CResource::streamSync()
{
    m_imp->streamSync();
}
void CResource::mapGLToCUDA()
{
    m_mapped = true ; 
    //printf("CResource::mapGLToCUDA %d\n", m_buffer_id);
    m_imp->registerBuffer();
    m_imp->mapGLToCUDA();
}
void CResource::unmapGLToCUDA()
{
    m_mapped = false ; 
    //printf("CResource::unmapGLToCUDA\n");
    m_imp->unmapGLToCUDA();
    m_imp->unregisterBuffer();
}



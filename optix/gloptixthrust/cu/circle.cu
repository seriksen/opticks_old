#include "hip/hip_runtime.h"
#include <optix_world.h>
using namespace optix;

rtBuffer<float4, 1>  vtx_buffer;

rtDeclareVariable(float, radius, , );
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );


RT_PROGRAM void circle_make_vertices()
{
    float frac = float(launch_index)/float(launch_dim) ; 
    float sinPhi, cosPhi;
    sincosf(2.f*M_PIf*frac,&sinPhi,&cosPhi);

    if(launch_index < 10)
        rtPrintf( "circle_make_vertices launch dim %d index %d radius %10.4f frac %10.4f s %10.4f c %10.4f \n", launch_dim, launch_index, radius, frac, sinPhi, cosPhi);

    vtx_buffer[launch_index] = make_float4( radius*sinPhi,  radius*cosPhi,  0.0f, 1.0f) ;
}


RT_PROGRAM void circle_dump()
{
    float4 v = vtx_buffer[launch_index] ; 
    if(launch_index < 10)
        rtPrintf( "circle_dump (dim,index) (%d,%d)  [%10.4f,%10.4f,%10.4f,%10.4f] \n", 
            launch_dim, launch_index, v.x, v.y, v.z, v.w  );
}




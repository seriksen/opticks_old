#include "hip/hip_runtime.h"
#include <optix_world.h>
using namespace optix;


// from optixrap/cu/helpers.h

// Convert a float3 in [0,1)^3 to a uchar4 in [0,255]^4 -- 4th channel is set to 255
static __device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.z)*255.99f),  // B 
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  // G 
                               static_cast<unsigned char>(__saturatef(c.x)*255.99f),  // R 
                               255u);                                                 // A 
}

/*
static __device__ __inline__ optix::uchar4 make_color(const optix::float4& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.z)*255.99f),  // B 
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  // G
                               static_cast<unsigned char>(__saturatef(c.x)*255.99f),  // R 
                               static_cast<unsigned char>(__saturatef(c.w)*255.99f));  // A
}
*/


struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int depth;
};




rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned,     radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable(rtObject,      top_object, , );

rtBuffer<uchar4, 2>   output_buffer;



rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );  
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );


RT_PROGRAM void raygen()
{

    PerRayData_radiance prd;
    prd.result = make_float3( 1.f, 0.f, 0.f ) ;

    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f ;

    optix::Ray ray = optix::make_Ray( eye, normalize(d.x*U + d.y*V + W), radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX) ; 
    rtTrace(top_object, ray, prd);

     //rtPrintf("//raygen launch_index.x %u launch_index.y %u launch_dim.x %u launch_dim.y %u \n", launch_index.x , launch_index.y, launch_dim.x , launch_dim.y   );
    output_buffer[launch_index] = make_color( prd.result ) ; 

    // make_uchar4(  255u, 0u, 0u,255u) ;  // red  (was expecting BGRA get RGBA)
}

// Returns shading normal as the surface shading result
RT_PROGRAM void closest_hit_radiance0()
{
  prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}

RT_PROGRAM void miss()
{
  prd_radiance.result = make_float3(1.f, 1.f, 1.f) ;
}



RT_PROGRAM void printTest0()
{
     unsigned long long index = launch_index.x ;
     rtPrintf("//printTest0 d:%d launch_index.x %u launch_index.y %u launch_dim.x %u launch_dim.y %u \n", index, launch_index.x , launch_index.y, launch_dim.x , launch_dim.y   );
}

RT_PROGRAM void printTest1()
{
     unsigned long long index = launch_index.x ;
     rtPrintf("//printTest1 llu:%llu launch_index.x %u launch_index.y %u launch_dim.x %u launch_dim.y %u \n", index, launch_index.x , launch_index.y, launch_dim.x , launch_dim.y   );
}



RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}



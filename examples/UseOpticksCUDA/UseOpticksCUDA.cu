#include <cstdio>
#include <vector>

#include "hip/hip_runtime.h"
#include "hip/driver_types.h"   // for hipError_t
#include "hip/hip_runtime_api.h"    // for _cudaGetErrorEnum


int main()
{
    printf(" CUDA_VERSION  %d \n", CUDA_VERSION ) ; 

    std::vector<hipError_t> errs ; 

    errs.push_back(hipSuccess); 
    errs.push_back(hipErrorLaunchFailure); 
    errs.push_back(hipErrorLaunchTimeOut); 
 
    for(unsigned i=0 ; i < errs.size() ; i++)
    {
        hipError_t err = errs[i] ; 
        const char* err_ = _cudaGetErrorEnum(err) ; 

        printf(" %4d %s \n", err, err_ ? err_ : "?" );
   }

    return 0 ; 
}

#include "TBuf.hh"

#include "strided_range.h"
#include "strided_repeated_range.h"

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iterator>
#include <iomanip>
#include <iostream>

#include "NPY.hpp"

template <typename T>
void TBuf::download(NPY<T>* npy)
{
    unsigned int numBytes = npy->getNumBytes(0) ;
    assert(numBytes == getNumBytes());
    void* src = getDevicePtr();
    void* dst = npy->zero();
    hipMemcpy( dst, src, numBytes, hipMemcpyDeviceToHost );
}


template <typename T>
void TBuf::dump(const char* msg, unsigned int stride, unsigned int begin, unsigned int end )
{
    Summary(msg);

    thrust::device_ptr<T> p = thrust::device_pointer_cast((T*)getDevicePtr()) ;

    if( stride == 0 )
    {
        thrust::copy( p + begin, p + end, std::ostream_iterator<T>(std::cout, " \n") );
    }
    else
    {
        typedef typename thrust::device_vector<T>::iterator Iterator;
        strided_range<Iterator> sri( p + begin, p + end, stride );
        thrust::copy( sri.begin(), sri.end(), std::ostream_iterator<T>(std::cout, " \n") );
    }
}


template <typename T>
void TBuf::dumpint(const char* msg, unsigned int stride, unsigned int begin, unsigned int end)
{

    // dumpint necessitated in addition to dump as streaming unsigned char gives characters not integers

    Summary(msg);

    thrust::device_ptr<T> p = thrust::device_pointer_cast((T*)getDevicePtr()) ;

    thrust::host_vector<T> h ;

    if( stride == 0 )
    {
        h.resize(thrust::distance(p+begin, p+end));
        thrust::copy( p + begin, p + end, h.begin());
    }
    else
    {
        typedef typename thrust::device_vector<T>::iterator Iterator;
        strided_range<Iterator> sri( p + begin, p + end, stride );
        h.resize(thrust::distance(sri.begin(), sri.end()));
        thrust::copy( sri.begin(), sri.end(), h.begin() );
    }

    for(unsigned int i=0 ; i < h.size() ; i++)
    {
        std::cout
                 << std::setw(7) << i
                 << std::setw(7) << int(h[i])
                 << std::endl ;
    }
}







template <typename T>
T TBuf::reduce(unsigned int stride, unsigned int begin, unsigned int end )
{
    thrust::device_ptr<T> p = thrust::device_pointer_cast((T*)getDevicePtr()) ;

    T result ;
    if( stride == 0 )
    {
        result = thrust::reduce( p + begin, p + end );
    }
    else
    {
        typedef typename thrust::device_vector<T>::iterator Iterator;
        strided_range<Iterator> sri( p + begin, p + end, stride );
        result = thrust::reduce( sri.begin(), sri.end() );
    }
    return result ;
}



template <typename T>
void TBuf::repeat_to( TBuf* other, unsigned int stride, unsigned int begin, unsigned int end, unsigned int repeats )
{
    thrust::device_ptr<T> src = thrust::device_pointer_cast((T*)getDevicePtr()) ;
    thrust::device_ptr<T> tgt = thrust::device_pointer_cast((T*)other->getDevicePtr()) ;

    typedef typename thrust::device_vector<T>::iterator Iterator;

    strided_repeated_range<Iterator> si( src + begin, src + end, stride, repeats);

    thrust::copy( si.begin(), si.end(),  tgt );    
}



template void TBuf::dump<unsigned int>(const char*, unsigned int, unsigned int, unsigned int);
template void TBuf::dumpint<unsigned char>(const char*, unsigned int, unsigned int, unsigned int);
template void TBuf::repeat_to<unsigned char>(TBuf*, unsigned int, unsigned int, unsigned int, unsigned int);
template unsigned int TBuf::reduce<unsigned int>(unsigned int, unsigned int, unsigned int);
template void TBuf::download<unsigned char>(NPY<unsigned char>* npy);




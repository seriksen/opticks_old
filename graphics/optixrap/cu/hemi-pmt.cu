#include "hip/hip_runtime.h"
// based on /usr/local/env/cuda/OptiX_380_sdk/julia/sphere.cu

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4,  sphere, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(unsigned int, instance_index,  ,);
rtDeclareVariable(unsigned int, primitive_count, ,);

rtBuffer<float4> partBuffer; 
rtBuffer<uint4>  solidBuffer; 
rtBuffer<uint4>  identityBuffer; 


// attributes communicate to closest hit program,
// they must be set inbetween rtPotentialIntersection and rtReportIntersection

rtDeclareVariable(uint4, instanceIdentity,   attribute instance_identity,);
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 



/*
Ericson, Real Time Collision Detection p196-198



   Ray

         L(t) = A + t(B - A)   


      A ----------------------------------B



   Cylinder

            d = Q - P               axis
            
            v = X - P               surface vec

                  v.d
            w =  ----- d            component of v along axis
                  d.d 

          r*r = (v - w).(v - w)     surface locus      


                                            B
                                           /
                    +--------Q--------+   /
                    |        |        |  /
                    |        |        | /
                    |        |        |/ 
                    |        |        * 
                    |        |       /| 
                    |        |      / |      
                    |        |     /  |
                    |     d  |    /   |         
                    |        |   /    |    
                    |        |  /     |       
                    |        | /      |
                    |        |/       | 
                    |        /.  .  . | .  .  .  .  .  . 
                    |       /|        |                
                    |      / |        |                .
                    |     /  |        | 
                    |    /   |        |                .
                    |   /    |        | 
                    |  /     |        |                .
                    | /      |        | 
                    |/       |        |                .
                    *        |        | 
                   /|        |        |                .
                  / |        |        | 
                 /  |        |        |               n.d
                /   +--------P--------+
          n    /            .|   r                     .
              /           .  
             /          .    |                         .
            /         .     
           /        .        |                         .
          /       . 
         /      .  m         |   m.d                   .
        /     .             
       /    .                |                         . 
      /   .  
     /  .                    |                         .
    / .    
   A .  .   .  .   .   .   . | .  .  .  .  .  .   .    .    




       v = L(t) - P 

         = (A - P) +  t(B - A)


      v  =   m + t n

      m  = A - P           ray origin in cylinder frame

      m.d                  axial coordinate of ray origin 

      n  = B - A           ray direction



                  v.d
            w =  ----- d            component of v along axis
                  d.d 


                  m.d + t n.d
            w =  -------------- d      
                      d.d 

          r*r = (v - w).(v - w)   

          r*r  = v.v + w.w - 2 v.w  


          v.v = ( m + t n ).(m + t n)

              = m.m + 2t m.n + t*t n.n 



   Intersection with P endcap plane 

       (X - P).d = 0 

       ( A + t (B - A) - P).d = 0 

       (  m + t n ).d = 0        =>   t = - m.d / n.d          

                when axial n in d direction          

                                     t  = - m.n / n.n    
      radial requirement 

         (m + t n).(m + t n) < rr 

         mm - rr + 2t m.n + t*t nn < 0 


   Intersection with Q endcap plane 

       (X - Q).d = 0      Q = d + P  

       (A + t (B - A) - Q).d = 0 
 
       ( A - P + t (B - A) - d ).d = 0 

       (  m + t n - d ).d = 0      =>    t = ( d.d - m.d ) / n.d

                when axial n in d direction          


      radial requirement 

         (m + t n - d).(m + t n - d) < rr 

         mm + tt nn + dd  






*/

static __device__
void intersect_ztubs(const float4& zrg, const float4& q0, const float4& q1, const uint4& identity )
{
    float3 position = make_float3( q0 ); 
    float radius = q0.w;
    float sizeZ = q1.x ; 

    float3 m = ray.origin - position ;
    float3 n = ray.direction ; 
    float3 d = make_float3(0.f, 0.f, sizeZ ); 

    float rr = radius*radius ; 
    float3 dnorm = normalize(d);

    float mm = dot(m, m) ; 
    float nn = dot(n, n) ; 
    float dd = dot(d, d) ;  
    float nd = dot(n, d) ;
    float md = dot(m, d) ;
    float mn = dot(m, n) ; 
    float k = mm - rr ; 

    // quadratic coefficients of t,     a tt + 2b t + c = 0 
    float a = dd*nn - nd*nd ;   
    float b = dd*mn - nd*md ;
    float c = dd*k - md*md ; 

    float disc = b*b-a*c;

    // axial ray 
    if(fabs(a) < 1e-6f)     
    {
        if(c > 0.f) return ; // ray starts and ends outside cylinder
        if(md < 0.f)         // ray origin on P side
        {
            float t = -mn/nn ;    // P endcap 
            if( rtPotentialIntersection(t) )
            {
                shading_normal = geometric_normal = -dnorm  ;  
                instanceIdentity = identity ; 
                rtReportIntersection(0);
            }
        } 
        else if(md > dd) // ray origin on Q side 
        {
            float t = (nd - mn)/nn ;  // Q endcap
            if( rtPotentialIntersection(t) )
            {
                shading_normal = geometric_normal = dnorm ; 
                instanceIdentity = identity ; 
                rtReportIntersection(0);
            }
        }
        else    // md 0:dd, ray origin inside 
        {
            if( nd > 0.f ) // ray along +d 
            {
                float t = -mn/nn ;    // P endcap from inside
                if( rtPotentialIntersection(t) )
                {
                    shading_normal = geometric_normal = dnorm  ;  
                    instanceIdentity = identity ; 
                    rtReportIntersection(0);
                }
            } 
            else   // ray along -d
            {
                float t = (nd - mn)/nn ;  // Q endcap from inside
                if( rtPotentialIntersection(t) )
                {
                    shading_normal = geometric_normal = -dnorm ; 
                    instanceIdentity = identity ; 
                    rtReportIntersection(0);
                }
            }
        }
        return ; 
    }

    if(disc > 0.0f)  // intersection with the infinite cylinder
    {
        float sdisc = sqrtf(disc);

        float root1 = (-b - sdisc)/a;     // what about other root ? 
        float ad1 = md + root1*nd ;        // axial coord of intersection point 
        float3 P1 = ray.origin + root1*ray.direction ;  

        if( ad1 > 0.f && ad1 < dd )  // intersection inside cylinder range
        {
            if( rtPotentialIntersection(root1) ) 
            {
                float3 N  = (P1 - position)/radius  ;  
                N.z = 0.f ; 

                rtPrintf("intersect_ztubs r %10.4f disc %10.4f sdisc %10.4f root1 %10.4f P %10.4f %10.4f %10.4f N %10.4f %10.4f \n", 
                    radius, disc, sdisc, root1, P1.x, P1.y, P1.z, N.x, N.y );

                shading_normal = geometric_normal = normalize(N) ;
                instanceIdentity = identity ; 
                rtReportIntersection(0);
            } 
        } 
        else if( ad1 < 0.f ) //  intersection outside cylinder on P side
        {
            if( nd <= 0.f ) return ; // ray direction away from endcap
            float t = -md/nd ;   // P endcap 
            float checkr = k + t*(2.f*mn + t*nn) ; // bracket typo in book 2*t*t makes no sense   
            if ( checkr < 0.f )
            {
                if( rtPotentialIntersection(t) )
                {
                    shading_normal = geometric_normal = -dnorm  ;  
                    instanceIdentity = identity ; 
                    rtReportIntersection(0);
                }
            } 
        } 
        else if( ad1 > dd  ) //  intersection outside cylinder on Q side
        {
            if( nd >= 0.f ) return ; // ray direction away from endcap
            float t = (dd-md)/nd ;   // Q endcap 
            float checkr = k + dd - 2.0f*md + t*(2.f*(mn-nd)+t*nn) ;             
            if ( checkr < 0.f )
            {
                if( rtPotentialIntersection(t) )
                {
                    shading_normal = geometric_normal = dnorm  ;  
                    instanceIdentity = identity ; 
                    rtReportIntersection(0);
                }
            } 
        }


        float root2 = (-b + sdisc)/a;     // far root : means are inside (always?)
        float ad2 = md + root2*nd ;        // axial coord of far intersection point 
        float3 P2 = ray.origin + root2*ray.direction ;  


        if( ad2 > 0.f && ad2 < dd )  // intersection from inside against wall 
        {
            if( rtPotentialIntersection(root2) ) 
            {
                float3 N  = (P2 - position)/radius  ;  
                N.z = 0.f ; 

                shading_normal = geometric_normal = -normalize(N) ;
                instanceIdentity = identity ; 
                rtReportIntersection(0);
            } 
        } 
        else if( ad2 < 0.f ) //  intersection from inside to P endcap
        {
            float t = -md/nd ;   // P endcap 
            float checkr = k + t*(2.f*mn + t*nn) ; // bracket typo in book 2*t*t makes no sense   
            if ( checkr < 0.f )
            {
                if( rtPotentialIntersection(t) )
                {
                    shading_normal = geometric_normal = dnorm  ;  
                    instanceIdentity = identity ; 
                    rtReportIntersection(0);
                }
            } 
        } 
        else if( ad2 > dd  ) //  intersection from inside to Q endcap
        {
            float t = (dd-md)/nd ;   // Q endcap 
            float checkr = k + dd - 2.0f*md + t*(2.f*(mn-nd)+t*nn) ;             
            if ( checkr < 0.f )
            {
                if( rtPotentialIntersection(t) )
                {
                    shading_normal = geometric_normal = -dnorm  ;  
                    instanceIdentity = identity ; 
                    rtReportIntersection(0);
                }
            } 
        }





    }
}

/*

    Ray-Sphere
    ~~~~~~~~~~~~~

    Ray(xyz) = ori + t*dir     dir.dir = 1

    (t*dir + ori-cen).(t*dir + ori-cen) = rad^2

     t^2 dir.dir + 2t(ori-cen).dir + (ori-cen).(ori-cen) - rad^2 = 0  

     t^2 + 2t O.D + O.O - radius = 0 

     a t^2 + b t + c = 0  =>   t = ( -b +- sqrt(b^2 - 4ac) )/2a 


        t = -2 O.D +-  sqrt(4* [(b/2*b/2) - (O.O - rad*rad)])
            ----------------------------------------- 
                            2

          =   - O.D +- sqrt(  O.D*O.D - (O.O - rad*rad) ) 


      normal to sphere at intersection point  (O + t D)/radius

            (ori + t D) - cen
            ------------------
                  radius


     Ray-Cylinder
     ~~~~~~~~~~~~~

         
          \
           \
     +----+-\--+
     |    |  \ |
     |    |   \|
     +----A----P--> normal   
     |    |   /|\
     |    |  / | \
     |    | /  |  \  ray        
     |    |/   |   \
     +----C----+    \
             r 
         
     Cylinder position C, axis C-A, intersection with ray at P

    
     Normal at intersection point is component of P-C 
     with the axial component subtracted

        (P-C) - (P-C).(A-C)


     float3 I = (P - position)/radius ; // cylinder intersection point in cylinder frame and radius units   



*/

template<bool use_robust_method>
static __device__
void intersect_zsphere(const float4& zrg, const float4& q0, const uint4& identity)
{
  float3 center = make_float3(q0);
  float radius = q0.w;

  float3 O = ray.origin - center;
  float3 D = ray.direction;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;

 /*
  rtPrintf("intersect_sphere %10.4f %10.4f %10.4f : %10.4f disc %10.4f \n", 
       center.x,  
       center.y,  
       center.z,  
       radius,
       disc);  
  */

    if(disc > 0.0f)
    {
        float sdisc = sqrtf(disc);
        float root1 = (-b - sdisc);
        float root11 = 0.0f;
        bool do_refine = use_robust_method && fabsf(root1) > 10.f * radius ;  // long ray 

        if(do_refine) // refine root1
        {
            float3 O1 = O + root1 * ray.direction;  //  move origin along to 1st intersection point
            b = dot(O1, D);
            c = dot(O1, O1) - radius*radius;
            disc = b*b - c;
            if(disc > 0.0f) 
            {
                sdisc = sqrtf(disc);
                root11 = (-b - sdisc);
            }
        }
        float3 P = ray.origin + (root1 + root11)*ray.direction ;  
        if( P.z > zrg.x && P.z < zrg.y )
        {
            bool check_second = true;
            if( rtPotentialIntersection( root1 + root11 ) ) 
            {
                shading_normal = geometric_normal = (O + (root1 + root11)*D)/radius;
                instanceIdentity = identity ; 
                if(rtReportIntersection(0)) check_second = false;
            } 
            if(check_second) 
            {
                float root2 = (-b + sdisc) + (do_refine ? root11 : 0.f);   // unconfirmed change root1 -> root11
                P = ray.origin + root2*ray.direction ;  
                if( P.z > zrg.x && P.z < zrg.y )
                { 
                    if( rtPotentialIntersection( root2 ) ) 
                    {
                        shading_normal = geometric_normal = (O + root2*D)/radius;
                        instanceIdentity = identity ; 
                        rtReportIntersection(0);   // material index 0 
                    }
                }
            }
        }
    }
}

static __device__
void intersect_aabb(const float4& q2, const float4& q3, const uint4& identity)
{
  const float3 min_ = make_float3(q2.x, q2.y, q2.z); 
  const float3 max_ = make_float3(q3.x, q3.y, q3.z); 

  float3 t0 = (min_ - ray.origin)/ray.direction;
  float3 t1 = (max_ - ray.origin)/ray.direction;

  // slab method 
  float3 near = fminf(t0, t1);
  float3 far = fmaxf(t0, t1);
  float tmin = fmaxf( near );
  float tmax = fminf( far );

  float3 n = make_float3(0.f);  

  if(tmin <= tmax) 
  {
      if(rtPotentialIntersection(tmin))
      {
          // hmm what about inside box ?
          if(     tmin == near.x) n.x = 1. ;
          else if(tmin == near.y) n.y = 1. ;
          else if(tmin == near.z) n.z = 1. ;

          shading_normal = geometric_normal = n ;
          instanceIdentity = identity ;
          rtReportIntersection(0);   // material index 0 
      } 
  }
}

static __device__
bool intersect_aabb(const float4& q2, const float4& q3)
{
    const float3 min_ = make_float3(q2.x, q2.y, q2.z); 
    const float3 max_ = make_float3(q3.x, q3.y, q3.z); 
    float3 t0 = (min_ - ray.origin)/ray.direction;
    float3 t1 = (max_ - ray.origin)/ray.direction;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    float tmin = fmaxf( near );
    float tmax = fminf( far );
    return tmin <= tmax ;
}



RT_PROGRAM void intersect(int primIdx)
{
  const uint4& solid    = solidBuffer[primIdx]; 
  unsigned int numParts = solid.y ; 
  const uint4& identity = identityBuffer[primIdx] ; 
  //const uint4 identity = identityBuffer[instance_index*primitive_count+primIdx] ;  // just primIdx for non-instanced


  for(unsigned int p=0 ; p < numParts ; p++)
  {  
      unsigned int partIdx = solid.x + p ;  

      const float4& q0 = partBuffer[4*partIdx+0];  
      const float4& q1 = partBuffer[4*partIdx+1];  
      const float4& q2 = partBuffer[4*partIdx+2] ;
      const float4& q3 = partBuffer[4*partIdx+3]; 

      float4 zrange = make_float4( q2.z , q3.z, 0.f, 0.f ) ;
      int typecode = __float_as_int(q2.w); 

      switch(typecode)
      {
          case 0:
                intersect_aabb(q2, q3, identity);
                break ; 
          case 1:
                intersect_zsphere<true>(zrange, q0, identity);
                break ; 
          case 2:
                //intersect_aabb(q2, q3, identity);
                intersect_ztubs(zrange,q0,q1, identity);
                break ; 
      }
  }

}





RT_PROGRAM void bounds (int primIdx, float result[6])
{
  // could do offline
  const uint4& solid    = solidBuffer[primIdx]; 
  unsigned int numParts = solid.y ; 

  optix::Aabb* aabb = (optix::Aabb*)result;
  *aabb = optix::Aabb();

  for(unsigned int p=0 ; p < numParts ; p++)
  { 
      unsigned int partIdx = solid.x + p ;  
      const float4& q2 = partBuffer[4*partIdx+2] ;
      const float4& q3 = partBuffer[4*partIdx+3]; 

      aabb->include( make_float3(q2), make_float3(q3) );
  } 

/*
  rtPrintf("bounds primIdx %d min %10.4f %10.4f %10.4f max %10.4f %10.4f %10.4f \n", primIdx, 
       result[0],
       result[1],
       result[2],
       result[3],
       result[4],
       result[5]
     );
*/

}


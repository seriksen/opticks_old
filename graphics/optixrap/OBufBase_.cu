#include "OBufBase.hh"
#include "NPYBase.hpp"

void OBufBase::init()
{
    examineBufferFormat(m_buffer->getFormat());
    m_size = getSize(m_buffer);
    m_numbytes = getNumBytes(m_buffer);
}

void OBufBase::examineBufferFormat(RTformat format)
{
   unsigned int mul(0) ;
   unsigned int soa(0) ;
   switch(format)
   {   
      case RT_FORMAT_UNKNOWN: mul=0 ;soa=0 ;  break ; 

      case RT_FORMAT_FLOAT:   mul=1 ; soa=sizeof(float) ; break ;
      case RT_FORMAT_FLOAT2:  mul=2 ; soa=sizeof(float) ; break ;
      case RT_FORMAT_FLOAT3:  mul=3 ; soa=sizeof(float) ; break ;
      case RT_FORMAT_FLOAT4:  mul=4 ; soa=sizeof(float) ; break ;

      case RT_FORMAT_BYTE:    mul=1 ; soa=sizeof(char)  ; break ;
      case RT_FORMAT_BYTE2:   mul=2 ; soa=sizeof(char)  ; break ;
      case RT_FORMAT_BYTE3:   mul=3 ; soa=sizeof(char)  ; break ;
      case RT_FORMAT_BYTE4:   mul=4 ; soa=sizeof(char)  ; break ;

      case RT_FORMAT_UNSIGNED_BYTE:  mul=1 ; soa=sizeof(unsigned char) ; break ;
      case RT_FORMAT_UNSIGNED_BYTE2: mul=2 ; soa=sizeof(unsigned char) ; break ;
      case RT_FORMAT_UNSIGNED_BYTE3: mul=3 ; soa=sizeof(unsigned char) ; break ;
      case RT_FORMAT_UNSIGNED_BYTE4: mul=4 ; soa=sizeof(unsigned char) ; break ;

      case RT_FORMAT_SHORT:  mul=1 ; soa=sizeof(short) ; break ;
      case RT_FORMAT_SHORT2: mul=2 ; soa=sizeof(short) ; break ;
      case RT_FORMAT_SHORT3: mul=3 ; soa=sizeof(short) ; break ;
      case RT_FORMAT_SHORT4: mul=4 ; soa=sizeof(short) ; break ;

      case RT_FORMAT_UNSIGNED_SHORT:  mul=1 ; soa=sizeof(unsigned short) ; break ;
      case RT_FORMAT_UNSIGNED_SHORT2: mul=2 ; soa=sizeof(unsigned short) ; break ;
      case RT_FORMAT_UNSIGNED_SHORT3: mul=3 ; soa=sizeof(unsigned short) ; break ;
      case RT_FORMAT_UNSIGNED_SHORT4: mul=4 ; soa=sizeof(unsigned short) ; break ;

      case RT_FORMAT_INT:  mul=1 ; soa=sizeof(int) ; break ;
      case RT_FORMAT_INT2: mul=2 ; soa=sizeof(int) ; break ;
      case RT_FORMAT_INT3: mul=3 ; soa=sizeof(int) ; break ;
      case RT_FORMAT_INT4: mul=4 ; soa=sizeof(int) ; break ;

      case RT_FORMAT_UNSIGNED_INT:  mul=1 ; soa=sizeof(unsigned int) ; break ;
      case RT_FORMAT_UNSIGNED_INT2: mul=2 ; soa=sizeof(unsigned int) ; break ;
      case RT_FORMAT_UNSIGNED_INT3: mul=3 ; soa=sizeof(unsigned int) ; break ;
      case RT_FORMAT_UNSIGNED_INT4: mul=4 ; soa=sizeof(unsigned int) ; break ;

      case RT_FORMAT_USER:       mul=0 ; soa=0 ; break ;
      case RT_FORMAT_BUFFER_ID:  mul=0 ; soa=0 ; break ;
      case RT_FORMAT_PROGRAM_ID: mul=0 ; soa=0 ; break ; 
   }   

    unsigned int element_size_bytes = getElementSizeInBytes(format);
    assert(element_size_bytes == soa*mul );

    setMultiplicity(mul)  ;
    setSizeOfAtom(soa) ;
}


void OBufBase::setSizeOfAtom(unsigned int soa)
{
    m_sizeofatom = soa ; 
} 
void OBufBase::setMultiplicity(unsigned int mul)
{
    m_multiplicity = mul ; 
} 


unsigned int OBufBase::getElementSizeInBytes(RTformat format)
{
    size_t element_size ; 
    rtuGetSizeForRTformat( format, &element_size);
    return element_size ; 
}

void* OBufBase::getDevicePtr()
{
    hipDeviceptr_t cu_ptr = m_buffer->getDevicePointer(m_device) ;
    return (void*)cu_ptr ; 
}

unsigned int OBufBase::getSize(const optix::Buffer& buffer)
{
    RTsize width, height, depth ; 
    buffer->getSize(width, height, depth);
    RTsize size = width*height*depth ; 
    return size ; 
}

unsigned int OBufBase::getNumBytes(const optix::Buffer& buffer)
{
    unsigned int size = getSize(buffer);

    RTformat format = buffer->getFormat() ;
    unsigned int element_size = getElementSizeInBytes(format);
    if(element_size == 0u && format == RT_FORMAT_USER)
    {
        element_size = buffer->getElementSize();
        printf("OBufBase::getNumBytes RT_FORMAT_USER element_size %u size %u \n", element_size, size );
    }
    return size*element_size ; 
}

void OBufBase::upload(NPYBase* npy)
{
    void* data = npy->getBytes() ;

    assert(data);

    unsigned int numBytes = npy->getNumBytes(0);

    printf("OBufBase::upload nbytes %u \n", numBytes);

    memcpy( m_buffer->map(), data, numBytes );

    m_buffer->unmap();
}


void OBufBase::download(NPYBase* npy)
{
    unsigned int numBytes = npy->getNumBytes(0) ;

    assert(numBytes == m_numbytes);

    void* ptr = m_buffer->map() ; 

    npy->read( ptr );

    m_buffer->unmap(); 
}


/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */


#include <iostream> 
#include <iomanip> 


#include <thrust/device_vector.h>
//#include <hiprand/hiprand_kernel.h> 

#include "SSys.hh"
#include "SStr.hh"
#include "BFile.hh"
#include "NPY.hpp"
#include "TRngBuf.hh"
#include "TUtil.hh"

#include "OPTICKS_LOG.hh"


int main(int argc, char** argv)
{
    OPTICKS_LOG(argc, argv);

    LOG(info) << argv[0] ;

    static const unsigned NI_DEFAULT = 100000 ; 
    static const unsigned IBASE = SSys::getenvint("TRngBuf_IBASE", 0) ; 
    static const unsigned NI = SSys::getenvint("TRngBuf_NI", NI_DEFAULT ); 
    static const unsigned NJ = 16 ; 
    static const unsigned NK = 16 ; 

    bool default_ni = NI == NI_DEFAULT ;  


    NPY<double>* ox = NPY<double>::make(NI, NJ, NK);

    ox->zero();

    thrust::device_vector<double> dox(NI*NJ*NK);

    CBufSpec spec = make_bufspec<double>(dox); 

    TRngBuf<double> trb(NI, NJ*NK, spec );

    trb.setIBase(IBASE) ; 

    trb.generate(); 

    trb.download<double>(ox, true) ; 


    const char* path = default_ni ? 
                                     SStr::Concat("$TMP/TRngBufTest_", IBASE, ".npy") 
                                  :
                                     SStr::Concat("$TMP/TRngBufTest_", IBASE, "_", NI, ".npy") 
                                  ; 


    LOG(info) << " save " << path ; 

    ox->save(path)  ;

    std::string spath = BFile::FormPath(path); 

    SSys::npdump(spath.c_str(), "np.float64", NULL, "suppress=True,precision=8" );

    hipDeviceSynchronize();  
}



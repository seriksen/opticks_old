
#include <iostream> 
#include <iomanip> 


#include <thrust/device_vector.h>
//#include <hiprand/hiprand_kernel.h> 

#include "SSys.hh"
#include "NPY.hpp"
#include "TRngBuf.hh"
#include "TUtil.hh"

#include "OPTICKS_LOG.hh"




int main(int argc, char** argv)
{
    OPTICKS_LOG(argc, argv);

    LOG(info) << argv[0] ;

    static const unsigned NI = 100000 ; 
    static const unsigned NJ = 16 ; 
    static const unsigned NK = 16 ; 

    NPY<double>* ox = NPY<double>::make(NI, NJ, NK);
    ox->zero();

    thrust::device_vector<double> dox(NI*NJ*NK);

    CBufSpec spec = make_bufspec<double>(dox); 

    TRngBuf<double> trb(NI, NJ*NK, spec );

    trb.generate(); 

    trb.download<double>(ox, true) ; 
  
    const char* path = "$TMP/TRngBufTest.npy" ; 
    //
    //  import os, numpy as np ; a = np.load(os.path.expandvars("$TMP/TRngBufTest.npy"))

    ox->save(path)  ;


    hipDeviceSynchronize();  
}


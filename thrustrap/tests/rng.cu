#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <hiprand/hiprand_kernel.h> 
#include <iostream>
#include <iomanip>

template<typename T>
struct PRNG
{
    typedef unsigned long long ULL ; 

    T*      dev ; 
    ULL    seed ; 
    ULL  offset ; 
    unsigned ni ; 
    unsigned nj ; 
    unsigned num_elem ; 

    __host__ 
    PRNG(T* dev_, unsigned ni_ , unsigned nj_,  ULL seed_=0ull , ULL offset_=0ull )
         : 
         dev(dev_),
         ni(ni_),
         nj(nj_),
         num_elem(ni*nj),
         seed(seed_), 
         offset(offset_)
         {}

    __device__
    void operator()(const unsigned uid ) const
    {
        hiprandState s;
        hiprand_init(seed, uid , offset, &s);

        for(unsigned j = 0; j < nj; ++j) 
        {   
            unsigned idx = uid*nj+j ;
            if(idx < num_elem )
            {   
                dev[idx] = hiprand_uniform(&s)  ;   
            }   
        }   
    }

    __host__
    void generate(unsigned i0, unsigned i1)
    {
        thrust::for_each(
              thrust::counting_iterator<unsigned>(i0),
              thrust::counting_iterator<unsigned>(i1),
               *this);
    }
};



int main(void)
{
    unsigned NI = 100 ;
    unsigned NJ = 16 ;
    unsigned N = NI*NJ ; 

    thrust::device_vector<float> dvec(N);
    float* udev = thrust::raw_pointer_cast(dvec.data()); 

    PRNG<float> prng(udev, NI, NJ ); 

    prng.generate(0, NI) ; 
    thrust::host_vector<float> hvec(dvec) ; 


    for( unsigned i=0 ; i < NI ; i++)
    {
        std::cout << std::setw(7) << i << " : " ; 
        for(unsigned j=0 ; j < NJ ; j++ )
             std::cout << " " << std::setw(10) << std::fixed << hvec[i*NJ+j] ; 
        std::cout << std::endl ; 
    }

    return 0;
}

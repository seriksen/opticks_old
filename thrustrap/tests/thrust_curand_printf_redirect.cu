#include "hip/hip_runtime.h"

// http://docs.nvidia.com/cuda/hiprand/device-api-overview.html#thrust-and-hiprand-example
// http://www.math.sci.hiroshima-u.ac.jp/~m-mat/MT/MTGP/mtgp3.pdf

#include <cassert> 
#include <cstdlib> 
#include <sstream> 

#include "SSys.hh"
#include "SDirect.hh"

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h> 
#include <hiprand/hiprand_kernel.h> 
#include <iostream> 
#include <iomanip> 


template <typename T>
struct curand_printf 
{ 
    T _seed ;
    T _offset ;
    T _seq0 ; 
    T _seq1 ; 
    T _zero ; 
    bool _logf ; 
 
    curand_printf( T seed , T offset, T seq0, T seq1, bool logf )
       :
       _seed(seed),
       _offset(offset),
       _seq0(seq0),
       _seq1(seq1),
       _logf(logf),
       _zero(0)
    {
    }
    
    __device__ 
    void operator()(unsigned id) 
    { 
        unsigned thread_offset = 0 ;  
        hiprandState s; 
        hiprand_init(_seed, id + thread_offset, _offset, &s); 
        printf(" id:%4u thread_offset:%u seq0:%llu seq1:%llu \n", id, thread_offset, _seq0, _seq1 );  
 
        for(T i = _zero ; i < _seq1 ; ++i) 
        { 
            float f = hiprand_uniform(&s); 
            if( i < _seq0 ) continue ; 

            printf(" %lf ", f );  

            if(_logf)
            {
                float lf = -logf(f)*1e7f ; 
                printf(" %lf ", lf );  

                //double d(f) ;   
                //double ld = -log(d)*1e7 ; 

                //double ld = -log(double(f))*1e7 ; 
                float ld = -log(double(f))*1e7 ; 
                printf(" %15.10g ", ld );  

            }

            if( i % 4 == 3 ) printf("\n") ; 
        } 





    } 
}; 

/*

__device__ void
hiprand_init (
    unsigned long long seed, 
    unsigned long long sequence,
    unsigned long long offset, 
    hiprandState_t *state)

The hiprand_init() function sets up an initial state allocated by the caller
using the given seed, sequence number, and offset within the sequence.
Different seeds are guaranteed to produce different starting states and
different sequences. The same seed always produces the same state and the same
sequence. The state set up will be the state after 2^67 sequence + offset calls
to hiprand() from the seed state.

*/


int main(int argc, char** argv) 
{ 
    int i0 = argc > 1 ? atoi(argv[1]) : 0 ; 
    int i1 = argc > 2 ? atoi(argv[2]) : i0+1 ; 
    int q0 = argc > 3 ? atoi(argv[3]) : 0 ; 
    int q1 = argc > 4 ? atoi(argv[4]) : 16 ; 

    char* LOGF = getenv("LOGF") ; 
    bool logf = LOGF != NULL ; 

    std::cout 
         << argv[0]
         << std::endl  
         << " i0 " << i0  
         << " i1 " << i1
         << " q0 " << q0  
         << " q1 " << q1
         << " logf " << ( logf ? "Y" : "N" ) 
         << std::endl 
         ; 

    assert( i0 >= 0 && i1 >= 0 );
    assert( q0 >= 0 && q1 >= 0 );
    assert( i0 < i1 );
    assert( q0 < q1 );

    SSys::Dump(argv[0]);

    std::stringstream coutbuf;
    std::stringstream cerrbuf;
    {   
        cout_redirect out_(coutbuf.rdbuf());
        cerr_redirect err_(cerrbuf.rdbuf());

        SSys::Dump(argv[0]);

        thrust::for_each( 
                thrust::counting_iterator<int>(i0), 
                thrust::counting_iterator<int>(i1), 
                curand_printf<unsigned long long>(0,0,q0,q1,logf));

        hipDeviceSynchronize();  

        // dtors of the redirect structs reset back to standard cout/cerr streams  
    }   

    // **THIS FAILS TO CAPTURE OUTPUT FROM THE KERNEL**
    //    thrust_curand_printf_redirect2.cu  succeeds 
    //    using a lower level approach with sysrap-/S_freopen_redirect.hh

    std::string out = coutbuf.str();
    std::string err = cerrbuf.str();

    std::cout << "out:\n" << out << std::endl ; 
    std::cout << "err:\n" << err << std::endl ; 

    SSys::Dump(argv[0]);


    return 0; 
} 


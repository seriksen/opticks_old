/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include "strided_range.h"

#include <thrust/fill.h>
#include <thrust/copy.h>
#include <ostream>

int main(void)
{
    thrust::device_vector<int> data(8);
    data[0] = 10;
    data[1] = 20;
    data[2] = 30;
    data[3] = 40;
    data[4] = 50;
    data[5] = 60;
    data[6] = 70;
    data[7] = 80;

    // print the initial data
    std::cout << "data: ";
    thrust::copy(data.begin(), data.end(), std::ostream_iterator<int>(std::cout, " "));  std::cout << std::endl;

    typedef thrust::device_vector<int>::iterator Iterator;
    
    // create strided_range with indices [0,2,4,6]
    strided_range<Iterator> evens(data.begin(), data.end(), 2);
    std::cout << "sum of even indices: " << thrust::reduce(evens.begin(), evens.end()) << std::endl;
    
    // create strided_range with indices [1,3,5,7]
    strided_range<Iterator> odds(data.begin() + 1, data.end(), 2);
    std::cout << "sum of odd indices:  " << thrust::reduce(odds.begin(), odds.end()) << std::endl;

    // set odd elements to 0 with fill()
    std::cout << "setting odd indices to zero: ";
    thrust::fill(odds.begin(), odds.end(), 0);
    thrust::copy(data.begin(), data.end(), std::ostream_iterator<int>(std::cout, " "));  std::cout << std::endl;

    return 0;
}
/*
simon:thrustrap blyth$ /usr/local/env/numerics/thrustrap/bin/strided_rangeTest
data: 10 20 30 40 50 60 70 80 
sum of even indices: 160
sum of odd indices:  200
setting odd indices to zero: 10 0 30 0 50 0 70 0 
*/



#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

#ifdef __clang__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wlanguage-extension-token"
#endif

#include "cuRANDWrapper_kernel.hh"
#include "LaunchSequence.hh"
#include "hiprand/hiprand_kernel.h"

#ifdef __clang__
#pragma clang diagnostic pop
#endif





#define CUDA_SAFE_CALL( call) do {                                         \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                      \
    } } while (0)




hipDeviceptr_t allocate_rng_wrapper( LaunchSequence* launchseq)
{
    unsigned int items = launchseq->getItems(); 
    size_t nbytes = items*sizeof(hiprandState) ;
    int value = 0 ; 

    hipDeviceptr_t dev_rng_states ;

    CUDA_SAFE_CALL( hipMalloc((void**)&dev_rng_states, nbytes )); 

    CUDA_SAFE_CALL( hipMemset((void*)dev_rng_states, value, nbytes ));
    
    return dev_rng_states ;
}


void free_rng_wrapper( hipDeviceptr_t dev_rng_states )
{
    CUDA_SAFE_CALL( hipFree((void*)dev_rng_states));
}



hiprandState* copytohost_rng_wrapper( LaunchSequence* launchseq, hipDeviceptr_t dev_rng_states)
{
    unsigned int items = launchseq->getItems(); 

    void* host_rng_states = malloc(sizeof(hiprandState)*items);

    CUDA_SAFE_CALL( hipMemcpy(host_rng_states, (void*)dev_rng_states, sizeof(hiprandState)*items, hipMemcpyDeviceToHost) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    return (hiprandState*)host_rng_states ;
}


hipDeviceptr_t copytodevice_rng_wrapper( LaunchSequence* launchseq, void* host_rng_states)
{
    // allocate device memory, copy from host, return device ptr

    unsigned int items = launchseq->getItems(); 

    hipDeviceptr_t dev_rng_states; 

    CUDA_SAFE_CALL( hipMalloc((void**)&dev_rng_states, items*sizeof(hiprandState))); 

    CUDA_SAFE_CALL( hipMemcpy((void*)dev_rng_states, host_rng_states, sizeof(hiprandState)*items, hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    return dev_rng_states ;
}








__global__ void init_rng(int threads_per_launch, int thread_offset, hiprandState* rng_states, unsigned long long seed, unsigned long long offset)
{
   // chroma approach is to recycle rng_states for each kernel launch 
   // in the cohort being propagated, which means the size of each kernel
   // launch is limited by timeouts occuring in any of the kernel launches 
   // including the hiprand_init one
   //
   // instead of doing this try having a state for every photon
   // and offsetting into it : the advantage is that changes to 
   // the CUDA launch configuration should not have any impact 
   // on the random number streams being consumed by the simulation (?) 
   //
   // But all these rng streams are rather expensive though, so 
   // should compare performace with chroma piecewise approach.
   // Maybe are just paying the expense at initialization ?
   //   

   int id = blockIdx.x*blockDim.x + threadIdx.x;
   if (id >= threads_per_launch) return;

   hiprand_init(seed, id + thread_offset , offset, &rng_states[id]);  

   // not &rng_states[id+thread_offset] as rng_states is offset already in kernel call
   //
   // hiprand_init runs 10x slower for large thread_offset ? starting from 262144
   // running the kernel launch sequence in reverse confirms this finding 
   //
   // :google:`hiprand_init slow with large sequence numbers`
   //
}



void before_kernel( hipEvent_t& start, hipEvent_t& stop )
{
    CUDA_SAFE_CALL( hipEventCreate( &start ) );
    CUDA_SAFE_CALL( hipEventCreate( &stop ) );
    CUDA_SAFE_CALL( hipEventRecord( start,0 ) );
}

void after_kernel( hipEvent_t& start, hipEvent_t& stop, float& kernel_time )
{
    CUDA_SAFE_CALL( hipEventRecord( stop,0 ) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop) );

    CUDA_SAFE_CALL( hipEventElapsedTime(&kernel_time, start, stop) );
    CUDA_SAFE_CALL( hipEventDestroy( start ) );
    CUDA_SAFE_CALL( hipEventDestroy( stop ) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}



void devicesync_wrapper()
{
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}


void init_rng_wrapper( LaunchSequence* launchseq, hipDeviceptr_t dev_rng_states, unsigned long long seed, unsigned long long offset)
{
    hipEvent_t start, stop ;

    for(unsigned int i=0 ; i < launchseq->getNumLaunches() ; i++ )
    {
        Launch& launch = launchseq->getLaunch(i) ;
    
        hiprandState* dev_rng_states_launch = (hiprandState*)dev_rng_states + launch.thread_offset ; 

        before_kernel( start, stop );

        init_rng<<<launch.blocks_per_launch, launch.threads_per_block>>>( launch.threads_per_launch, launch.thread_offset, dev_rng_states_launch, seed, offset );

        after_kernel( start, stop, launch.kernel_time );

        launch.Summary("init_rng_wrapper");
    } 
    launchseq->Summary("init_rng_wrapper");
}






__global__ void test_rng(int threads_per_launch, int thread_offset, hiprandState* rng_states, float *a)
{ 
   //

    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= threads_per_launch) return;

    // NB no id offsetting on rng_states or a, as the offsetting
    // was done once in the kernel call 
    // this means thread_offset argument not used

    hiprandState rng = rng_states[id];   // copy from global to register
    
    a[id] = hiprand_uniform(&rng);   

    rng_states[id] = rng;            // update from register to global

    //
    // hiprandState struct contains
    //        double boxmuller_extra_double
    // that causes demoting to float warnings
    // for the two above lines. 
    // Stanley Seibert judges it to be benign,
    //
    //   http://lists.tiker.net/pipermail/pycuda/2011-December/003513.html   
    //   
}



void test_rng_wrapper(
    LaunchSequence* launchseq,
    hipDeviceptr_t dev_rng_states,
    float* host_a
)
{
    hipEvent_t start, stop ;

    unsigned int items = launchseq->getItems(); 

    float* dev_a; 
    CUDA_SAFE_CALL(hipMalloc((void**)&dev_a, items*sizeof(float)));

    for(unsigned int i=0 ; i < launchseq->getNumLaunches() ; i++ )
    {
        Launch& launch = launchseq->getLaunch(i) ;

        hiprandState* dev_rng_states_launch = (hiprandState*)dev_rng_states + launch.thread_offset ; 
        float*       dev_a_launch = dev_a + launch.thread_offset ; 

        before_kernel( start, stop );

        test_rng<<<launch.blocks_per_launch, launch.threads_per_block>>>( launch.threads_per_launch, launch.thread_offset, dev_rng_states_launch, dev_a_launch );

        after_kernel( start, stop, launch.kernel_time );
    } 

    CUDA_SAFE_CALL( hipMemcpy(host_a, dev_a, items*sizeof(float), hipMemcpyDeviceToHost) ); 

    CUDA_SAFE_CALL( hipFree(dev_a) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}












/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include "OBufPair.hh"
#include "OBuf.hh"

#include "strided_range.h"
#include "iexpand.h"

template <typename T>
OBufPair<T>::OBufPair(CBufSlice src, CBufSlice dst ) 
   :
   m_src(src),
   m_dst(dst)
{
}


template <typename T>
void OBufPair<T>::seedDestination()
{
    typedef typename thrust::device_vector<T>::iterator Iterator;

    thrust::device_ptr<T> psrc = thrust::device_pointer_cast((T*)m_src.dev_ptr) ; 
    thrust::device_ptr<T> pdst = thrust::device_pointer_cast((T*)m_dst.dev_ptr) ; 

    strided_range<Iterator> si( psrc + m_src.begin, psrc + m_src.end, m_src.stride );
    strided_range<Iterator> di( pdst + m_dst.begin, pdst + m_dst.end, m_dst.stride );

    iexpand( si.begin(), si.end(), di.begin(), di.end() );

#ifdef DEBUG
    std::cout << "OBufPair<T>::seedDestination " << std::endl ; 
    thrust::copy( di.begin(), di.end(), std::ostream_iterator<T>(std::cout, " ") ); 
    std::cout << "OBufPair<T>::seedDestination " << std::endl ; 
#endif

}


template class OXRAP_API OBufPair<unsigned int> ;


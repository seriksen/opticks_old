#include "hip/hip_runtime.h"

// https://devtalk.nvidia.com/default/topic/574078/?comment=3896854

#include <hip/hip_vector_types.h>

#include "OBuf.hh"

#include "strided_range.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <iterator>
#include <iomanip>
#include <iostream>

__host__ std::ostream& operator<< (std::ostream& os, const optix::float4& p) 
{
        os << "[ " 
           << std::setw(10) << p.x << " " 
           << std::setw(10) << p.y << " "
           << std::setw(10) << p.z << " "
           << std::setw(10) << p.w << " "
           << " ]";
         return os;
}

__host__ std::ostream& operator<< (std::ostream& os, const optix::uint4& p) 
{
        os << "[ " 
           << std::setw(10) << p.x << " " 
           << std::setw(10) << p.y << " "
           << std::setw(10) << p.z << " "
           << std::setw(10) << p.w << " "
           << " ]";
         return os;
}

__host__ std::ostream& operator<< (std::ostream& os, const unsigned char& p) 
{
        os << " 0x" << std::hex << int(p) << std::dec << " " ;  
        return os;
}



OBuf::OBuf(const char* name, optix::Buffer& buffer, NPYBase* npy ) : OBufBase(name, buffer, npy)
{
}



template <typename T>
void OBuf::dump(const char* msg, unsigned int stride, unsigned int begin, unsigned int end)
{
    Summary(msg);

    thrust::device_ptr<T> p = thrust::device_pointer_cast((T*)getDevicePtr()) ; 
    if(m_hexdump) std::cout << std::hex ; 

    if( stride == 0 )
    {
        thrust::copy( p + begin, p + end, std::ostream_iterator<T>(std::cout, " \n") ); 
    }
    else
    {
        typedef typename thrust::device_vector<T>::iterator Iterator;
        strided_range<Iterator> sri( p + begin, p + end, stride );
        thrust::copy( sri.begin(), sri.end(), std::ostream_iterator<T>(std::cout, " \n") ); 
    }
    if(m_hexdump) std::cout << std::dec ; 
}


template <typename T>
void OBuf::dumpint(const char* msg, unsigned int stride, unsigned int begin, unsigned int end)
{

    // dumpint necessitated in addition to dump as streaming unsigned char gives characters not integers

    Summary(msg);

    thrust::device_ptr<T> p = thrust::device_pointer_cast((T*)getDevicePtr()) ; 


    thrust::host_vector<T> h ; 

    if( stride == 0 )
    {
        h.resize(thrust::distance(p+begin, p+end)); 
        thrust::copy( p + begin, p + end, h.begin()); 
    }
    else
    {
        typedef typename thrust::device_vector<T>::iterator Iterator;
        strided_range<Iterator> sri( p + begin, p + end, stride );
        h.resize(thrust::distance(sri.begin(), sri.end())); 
        thrust::copy( sri.begin(), sri.end(), h.begin() ); 
    }

    for(unsigned int i=0 ; i < h.size() ; i++)
    {
        std::cout 
                 << std::setw(7) << i 
                 << std::setw(7) << int(h[i])
                 << std::endl ;  
    }
}







template <typename T>
T OBuf::reduce(unsigned int stride, unsigned int begin, unsigned int end )
{
    // hmm this assumes do not do reductions at float4 level ?
    if(end == 0u) end = getNumAtoms(); 

    thrust::device_ptr<T> p = thrust::device_pointer_cast((T*)getDevicePtr()) ; 

    T result ; 
    if( stride == 0 )
    {
        result = thrust::reduce( p + begin, p + end ); 
    }
    else
    {
        typedef typename thrust::device_vector<T>::iterator Iterator;
        strided_range<Iterator> sri( p + begin, p + end, stride );
        result = thrust::reduce( sri.begin(), sri.end() ); 
    }
    return result ; 
}


/*
template optix::float4* OBuf::getDevicePtr<optix::float4>();
template optix::uint4* OBuf::getDevicePtr<optix::uint4>();
template unsigned int* OBuf::getDevicePtr<unsigned int>();
template unsigned long long* OBuf::getDevicePtr<unsigned long long>();
*/

template OXRAP_API void OBuf::dump<optix::float4>(const char*, unsigned int, unsigned int, unsigned int);
template OXRAP_API void OBuf::dump<optix::uint4>(const char*, unsigned int, unsigned int, unsigned int);
template OXRAP_API void OBuf::dump<unsigned int>(const char*, unsigned int, unsigned int, unsigned int);
template OXRAP_API void OBuf::dump<unsigned long long>(const char*, unsigned int, unsigned int, unsigned int);
template OXRAP_API void OBuf::dump<unsigned char>(const char*, unsigned int, unsigned int, unsigned int);
template OXRAP_API void OBuf::dump<int>(const char*, unsigned int, unsigned int, unsigned int);

template OXRAP_API void OBuf::dumpint<unsigned char>(const char*, unsigned int, unsigned int, unsigned int);


template OXRAP_API unsigned int OBuf::reduce<unsigned int>(unsigned int, unsigned int, unsigned int);
template OXRAP_API unsigned long long OBuf::reduce<unsigned long long>(unsigned int, unsigned int, unsigned int);




#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include <optix_world.h>
rtBuffer<rtCallableProgramId<unsigned(double,double,double,double*,unsigned)> > solve_callable ;

#include "cu/quad.h"
#include "cu/bbox.h"

typedef double Solve_t ;
#include "cu/Solve.h"

#define CSG_INTERSECT_TORUS_TEST 1
typedef double Torus_t ;
#include "cu/csg_intersect_torus.h"


using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4>  output_buffer;

RT_PROGRAM void iaTorusTest()
{
    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 

#ifdef CSG_INTERSECT_TORUS_TEST
    rtPrintf("// iaTorusTest %llu\n", photon_id);
#endif



    //  calling the below double laden function twice is prone to segv in createPTXFromFile
    //csg_intersect_torus_scale_test(photon_id, false);
    csg_intersect_torus_scale_test(photon_id, true );

    
    output_buffer[photon_offset+0] = make_float4(40.f, 40.f, 40.f, 40.f);
    output_buffer[photon_offset+1] = make_float4(41.f, 41.f, 41.f, 41.f);
    output_buffer[photon_offset+2] = make_float4(42.f, 42.f, 42.f, 42.f);
    output_buffer[photon_offset+3] = make_float4(43.f, 43.f, 43.f, 43.f);
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();

    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 
    
    output_buffer[photon_offset+0] = make_float4(-40.f, -40.f, -40.f, -40.f);
    output_buffer[photon_offset+1] = make_float4(-41.f, -41.f, -41.f, -41.f);
    output_buffer[photon_offset+2] = make_float4(-42.f, -42.f, -42.f, -42.f);
    output_buffer[photon_offset+3] = make_float4(-43.f, -43.f, -43.f, -43.f);
}


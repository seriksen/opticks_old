#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
//rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<float4>  output_buffer;

RT_PROGRAM void minimal()
{
    /*
    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 
    rtPrintf("// minimal %llu \n", photon_id );
    */ 

    unsigned photon_id = launch_index.x ;  
    unsigned photon_offset = photon_id*4 ; 
 
    rtPrintf("// minimalTest.cu:minimal %d \n", photon_id );
   
    output_buffer[photon_offset+0] = make_float4(40.f, 40.f, 40.f, 40.f);
    output_buffer[photon_offset+1] = make_float4(41.f, 41.f, 41.f, 41.f);
    output_buffer[photon_offset+2] = make_float4(42.f, 42.f, 42.f, 42.f);
    output_buffer[photon_offset+3] = make_float4(43.f, 43.f, 43.f, 43.f);

}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();

/*
    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 
    
    output_buffer[photon_offset+0] = make_float4(-40.f, -40.f, -40.f, -40.f);
    output_buffer[photon_offset+1] = make_float4(-41.f, -41.f, -41.f, -41.f);
    output_buffer[photon_offset+2] = make_float4(-42.f, -42.f, -42.f, -42.f);
    output_buffer[photon_offset+3] = make_float4(-43.f, -43.f, -43.f, -43.f);
*/

}




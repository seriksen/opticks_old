#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include <optix_world.h>

using namespace optix;

rtTextureSampler<float4, 2>  boundary_texture ;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<float4,2>       out_buffer;


RT_PROGRAM void boundaryTest()
{
    int ix = int(launch_index.x) ; 
    int iy = int(launch_index.y) ; 

    float x = (float(ix)+0.5f)/float(launch_dim.x) ; 
    float y = (float(iy)+0.5f)/float(launch_dim.y) ; 
    
    float4 val = tex2D(boundary_texture, x, y );

    //rtPrintf("boundaryTest (%d,%d) (%10.4f,%10.4f) -> (%10.4f,%10.4f,%10.4f,%10.4f)  \n", ix, iy, x, y, val.x, val.y, val.z, val.w);

    out_buffer[launch_index] = val ; 
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}




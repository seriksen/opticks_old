#include "hip/hip_runtime.h"

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

#include "cu/boundary_lookup.h"

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtBuffer<float4,2>  out_buffer;


RT_PROGRAM void boundaryLookupTest_ijk()
{
    int w = int(launch_index.x) ;
    int i = int(launch_index.y) ;
    float nm = boundary_domain.x + w*boundary_domain.z ;

    uint2 out_index ; 
    out_index.x = w ;  

    for(unsigned j=0 ; j < BOUNDARY_NUM_MATSUR ; j++){
    for(unsigned k=0 ; k < BOUNDARY_NUM_FLOAT4 ; k++)
    {
        out_index.y = boundary_lookup_ijk(i, j, k );  
        out_buffer[out_index] = boundary_lookup(nm, i, j, k ) ; 
    }
    }
}

RT_PROGRAM void boundaryLookupTest()
{
    int w = int(launch_index.x) ;
    int i = int(launch_index.y) ;
    float nm = boundary_domain.x + w*boundary_domain.z ;

    uint2 out_index ; 
    out_index.x = w ;  

    unsigned nj = BOUNDARY_NUM_MATSUR ;
    unsigned nk = BOUNDARY_NUM_FLOAT4 ;

    for(unsigned j=0 ; j < nj ; j++)
    {
        unsigned line = i*nj + j ;  
        for(unsigned k=0 ; k < nk ; k++)
        {
           out_index.y = boundary_lookup_linek(line, k );  
           out_buffer[out_index] = boundary_lookup(nm, line, k ) ; 
        }
    }
}


RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}

/*

       #float4
            |     ___ wavelength samples
            |    /
   (123, 4, 2, 39, 4)
    |    |          \___ float4 props        
  #bnd   | 
         |
    omat/osur/isur/imat  
*/



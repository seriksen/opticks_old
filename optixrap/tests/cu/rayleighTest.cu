#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<hiprandState, 1>       rng_states ;

rtBuffer<float4,2>  rayleigh_buffer;


// HMM : NOT GOOD THAT SO MUCH BAGGAGE NEEDED TO TEST rayleigh.h

#include "cu/quad.h"
#include "cu/boundary_lookup.h"

rtBuffer<uint4>                optical_buffer; 

#include "cu/state.h"
#include "cu/photon.h"
#include "cu/rayleigh.h"


RT_PROGRAM void rayleighTest()
{
    unsigned long long generate_id = launch_index.x ;
    //unsigned long long generate_offset = unsigned(generate_id)*4 ;
    hiprandState rng = rng_states[generate_id];

    Photon o, p ; 
    o.direction.x = 1.f ; 
    o.direction.y = 0.f ; 
    o.direction.z = 0.f ; 

    o.polarization.x = 0.f ; 
    o.polarization.y = 1.f ; 
    o.polarization.z = 0.f ; 

    p.direction.x = o.direction.x ; 
    p.direction.y = o.direction.y ; 
    p.direction.z = o.direction.z ; 

    p.polarization.x = o.polarization.x ; 
    p.polarization.y = o.polarization.y ; 
    p.polarization.z = o.polarization.z ; 
 
    rayleigh_scatter( p, rng );

    rtPrintf("//rayleighTest generate_id %llu \n", generate_id ); 

    uint2 u0 = make_uint2( unsigned(generate_id), 0u ) ;
    uint2 u1 = make_uint2( unsigned(generate_id), 1u ) ;
    uint2 u2 = make_uint2( unsigned(generate_id), 2u ) ;
    uint2 u3 = make_uint2( unsigned(generate_id), 3u ) ;

    rayleigh_buffer[u0] = make_float4( o.direction.x,     o.direction.y,     o.direction.z,     0.f );
    rayleigh_buffer[u1] = make_float4( o.polarization.x,  o.polarization.y,  o.polarization.z,  0.f );
    rayleigh_buffer[u2] = make_float4( p.direction.x,     p.direction.y,     p.direction.z,     0.f );
    rayleigh_buffer[u3] = make_float4( p.polarization.x,  p.polarization.y,  p.polarization.z,  0.f );
}


RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}



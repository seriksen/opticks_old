#include "hip/hip_runtime.h"
#include "NPYBase.hpp"
#include "OBufBase.hh"

OBufBase::OBufBase(const char* name, optix::Buffer& buffer) 
   :
   m_buffer(buffer), 
   m_name(strdup(name)), 
   m_multiplicity(0u), 
   m_sizeofatom(0u), 
   m_device(0u),
   m_hexdump(false)
{
    init();
}

OBufBase::~OBufBase()
{
   // no owned resources worth clearing up
}


CBufSlice OBufBase::slice( unsigned int stride, unsigned int begin, unsigned int end )
{
   return CBufSlice( getDevicePtr(), getSize(), getNumBytes(), stride, begin, end == 0u ? getNumAtoms() : end);
}

CBufSpec OBufBase::bufspec()
{
   return CBufSpec( getDevicePtr(), getSize(), getNumBytes()) ;
}


void OBufBase::Summary(const char* msg)
{
    printf("%s name %s size %u multiplicity %u sizeofatom %u NumAtoms %u NumBytes %u \n", 
         msg, 
         m_name, 
         getSize(), 
         m_multiplicity, 
         m_sizeofatom, 
         getNumAtoms(), 
         getNumBytes() );
}

void OBufBase::setHexDump(bool hexdump)
{
   m_hexdump = hexdump ; 
}

/*
   *getSize()*  Excludes multiplicity of the type of the OptiX buffer, ie the size
                is the number of float4 

         Examples:

          1) Cerenkov genstep NPY<float> buffer with dimensions (7836,6,4)
             is canonically represented as an OptiX float4 buffer of size 7836*6 = 47016 

          2) Torch genstep NPY<float> buffer with dimensions (1,6,4)
             is canonically represented as an OptiX float4 buffer of size 1*6 = 6 

*/


unsigned int OBufBase::getSize()  
{
    return getSize(m_buffer) ; 
}


unsigned int OBufBase::getMultiplicity()
{
    return m_multiplicity ; 
}
unsigned int OBufBase::getNumAtoms()
{
    return getSize()*m_multiplicity ; 
}
unsigned int OBufBase::getSizeOfAtom()
{
    return m_sizeofatom ; 
}
unsigned int OBufBase::getNumBytes()
{
    return getNumBytes(m_buffer) ; 
}

void OBufBase::init()
{
    examineBufferFormat(m_buffer->getFormat());
}

void OBufBase::examineBufferFormat(RTformat format)
{
   unsigned int mul(0) ;
   unsigned int soa(0) ;
   switch(format)
   {   
      case RT_FORMAT_UNKNOWN: mul=0 ;soa=0 ;  break ; 

      case RT_FORMAT_FLOAT:   mul=1 ; soa=sizeof(float) ; break ;
      case RT_FORMAT_FLOAT2:  mul=2 ; soa=sizeof(float) ; break ;
      case RT_FORMAT_FLOAT3:  mul=3 ; soa=sizeof(float) ; break ;
      case RT_FORMAT_FLOAT4:  mul=4 ; soa=sizeof(float) ; break ;

      case RT_FORMAT_BYTE:    mul=1 ; soa=sizeof(char)  ; break ;
      case RT_FORMAT_BYTE2:   mul=2 ; soa=sizeof(char)  ; break ;
      case RT_FORMAT_BYTE3:   mul=3 ; soa=sizeof(char)  ; break ;
      case RT_FORMAT_BYTE4:   mul=4 ; soa=sizeof(char)  ; break ;

      case RT_FORMAT_UNSIGNED_BYTE:  mul=1 ; soa=sizeof(unsigned char) ; break ;
      case RT_FORMAT_UNSIGNED_BYTE2: mul=2 ; soa=sizeof(unsigned char) ; break ;
      case RT_FORMAT_UNSIGNED_BYTE3: mul=3 ; soa=sizeof(unsigned char) ; break ;
      case RT_FORMAT_UNSIGNED_BYTE4: mul=4 ; soa=sizeof(unsigned char) ; break ;

      case RT_FORMAT_SHORT:  mul=1 ; soa=sizeof(short) ; break ;
      case RT_FORMAT_SHORT2: mul=2 ; soa=sizeof(short) ; break ;
      case RT_FORMAT_SHORT3: mul=3 ; soa=sizeof(short) ; break ;
      case RT_FORMAT_SHORT4: mul=4 ; soa=sizeof(short) ; break ;

      case RT_FORMAT_UNSIGNED_SHORT:  mul=1 ; soa=sizeof(unsigned short) ; break ;
      case RT_FORMAT_UNSIGNED_SHORT2: mul=2 ; soa=sizeof(unsigned short) ; break ;
      case RT_FORMAT_UNSIGNED_SHORT3: mul=3 ; soa=sizeof(unsigned short) ; break ;
      case RT_FORMAT_UNSIGNED_SHORT4: mul=4 ; soa=sizeof(unsigned short) ; break ;

      case RT_FORMAT_INT:  mul=1 ; soa=sizeof(int) ; break ;
      case RT_FORMAT_INT2: mul=2 ; soa=sizeof(int) ; break ;
      case RT_FORMAT_INT3: mul=3 ; soa=sizeof(int) ; break ;
      case RT_FORMAT_INT4: mul=4 ; soa=sizeof(int) ; break ;

      case RT_FORMAT_UNSIGNED_INT:  mul=1 ; soa=sizeof(unsigned int) ; break ;
      case RT_FORMAT_UNSIGNED_INT2: mul=2 ; soa=sizeof(unsigned int) ; break ;
      case RT_FORMAT_UNSIGNED_INT3: mul=3 ; soa=sizeof(unsigned int) ; break ;
      case RT_FORMAT_UNSIGNED_INT4: mul=4 ; soa=sizeof(unsigned int) ; break ;

      case RT_FORMAT_USER:       mul=0 ; soa=0 ; break ;
      case RT_FORMAT_BUFFER_ID:  mul=0 ; soa=0 ; break ;
      case RT_FORMAT_PROGRAM_ID: mul=0 ; soa=0 ; break ; 

#if OPTIX_VERSION >= 4000
      case RT_FORMAT_HALF  : mul=1 ; soa=sizeof(float)/2 ; break ; 
      case RT_FORMAT_HALF2 : mul=2 ; soa=sizeof(float)/2 ; break ; 
      case RT_FORMAT_HALF3 : mul=3 ; soa=sizeof(float)/2 ; break ; 
      case RT_FORMAT_HALF4 : mul=4 ; soa=sizeof(float)/2 ; break ; 
#endif

   }   

    unsigned int element_size_bytes = getElementSizeInBytes(format);
    assert(element_size_bytes == soa*mul );

    setMultiplicity(mul)  ;
    setSizeOfAtom(soa) ;
    // these do not change when buffer size changes
}


void OBufBase::setSizeOfAtom(unsigned int soa)
{
    m_sizeofatom = soa ; 
} 
void OBufBase::setMultiplicity(unsigned int mul)
{
    m_multiplicity = mul ; 
} 


unsigned int OBufBase::getElementSizeInBytes(RTformat format)
{
    size_t element_size ; 
    rtuGetSizeForRTformat( format, &element_size);
    return element_size ; 
}

void* OBufBase::getDevicePtr()
{
    //printf("OBufBase::getDevicePtr %s \n", ( m_name ? m_name : "-") ) ;
    //return (void*) m_buffer->getDevicePointer(m_device); 

    hipDeviceptr_t cu_ptr = (hipDeviceptr_t)m_buffer->getDevicePointer(m_device) ;
    return (void*)cu_ptr ; 
}

unsigned int OBufBase::getSize(const optix::Buffer& buffer)
{
    RTsize width, height, depth ; 
    buffer->getSize(width, height, depth);
    RTsize size = width*height*depth ; 
    return size ; 
}

unsigned int OBufBase::getNumBytes(const optix::Buffer& buffer)
{
    unsigned int size = getSize(buffer);

    RTformat format = buffer->getFormat() ;
    unsigned int element_size = getElementSizeInBytes(format);
    if(element_size == 0u && format == RT_FORMAT_USER)
    {
        element_size = buffer->getElementSize();
        printf("OBufBase::getNumBytes RT_FORMAT_USER element_size %u size %u \n", element_size, size );
    }
    return size*element_size ; 
}

void OBufBase::upload(NPYBase* npy)
{
    void* data = npy->getBytes() ;
    assert(data);

    unsigned int numBytes = npy->getNumBytes(0);
    unsigned int x_numBytes = getNumBytes();
    assert(numBytes == x_numBytes);

    printf("OBufBase::upload nbytes %u \n", numBytes);

    memcpy( m_buffer->map(), data, numBytes );

    m_buffer->unmap();
}


void OBufBase::download(NPYBase* npy)
{
    unsigned int numBytes = npy->getNumBytes(0) ;
    unsigned int x_numBytes = getNumBytes();
    assert(numBytes == x_numBytes);

    void* ptr = m_buffer->map() ; 

    npy->read( ptr );

    m_buffer->unmap(); 
}


#include "hip/hip_runtime.h"
#include "switches.h"

#include <optix.h>
#include <optix_math.h>

#include "PerRayData_radiance.h"

//geometric_normal is set by the closest hit intersection program 
rtDeclareVariable(float3, geometricNormal, attribute geometric_normal, );
rtDeclareVariable(uint4,  instanceIdentity, attribute instance_identity, );

rtDeclareVariable(float3, contrast_color, , );

rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t,            rtIntersectionDistance, );

rtDeclareVariable(unsigned int,  touch_mode, , );
rtDeclareVariable(float4,        ZProj, , );
rtDeclareVariable(float3,        front, , );
rtDeclareVariable(unsigned int,  parallel, , );


RT_PROGRAM void closest_hit_radiance()
{
    const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal)) ; 
    const float cos_theta = dot(n,ray.direction);

    float intensity = 0.5f*(1.0f-cos_theta) ;  // lambertian 

    float zHit_eye = -t*dot(front, ray.direction) ;   // intersect z coordinate (eye frame), always -ve 
    float zHit_ndc = parallel == 0 ? -ZProj.z - ZProj.w/zHit_eye : ZProj.z*zHit_eye + ZProj.w ;  // should be in range -1:1 for visibles
    float zHit_clip = 0.5f*zHit_ndc + 0.5f ;   // 0:1 for visibles

    //rtPrintf("closest_hit_radiance t %10.4f zHit_eye %10.4f  ZProj.z %10.4f ZProj.w %10.4f zHit_ndc %10.4f zHit_clip %10.4f \n", t, zHit_eye, ZProj.z, ZProj.w , zHit_ndc, zHit_clip );

    prd.result = make_float4(intensity, intensity, intensity, zHit_clip ); // hijack alpha for the depth 

#ifdef BOOLEAN_DEBUG
     switch(instanceIdentity.x)
     {
        case 1: prd.result.x = 1.f ; break ;
        case 2: prd.result.y = 1.f ; break ;
        case 3: prd.result.z = 1.f ; break ;
    }
#endif    

    prd.flag   = instanceIdentity.y ;   //  hijacked to become the hemi-pmt intersection code
}



// const float3 n = normalize(rtTransformNormal(RT_WORLD_TO_OBJECT, geometricNormal)) ; 
// const float3 n = normalize(geometricNormal) ; 
// currently lambertian from all the above three looks the same
 /*
  if(touch_mode)
  {
      // n.z often coming out very small, ~1e-9 
      //     this is just due to there being lots of vertical surfaces
      //     so the surface normal has no up-down component
      // 
      //  click on a PMT, and the many triangles oriented in all directions will give appropriate normals
      //
      rtPrintf("(touch)material1_radiance.cu geometricNormal  %10.4f %10.4f %10.4f   n %10.4f %10.4f %10.4f  ct %10.4f  \n",
          geometricNormal.x, 
          geometricNormal.y, 
          geometricNormal.z, 
          n.x, 
          n.y, 
          n.z,
          cos_theta  );
      //wavelength_check();
  }
  */

// normal shader colors dont match what getting with OpenGL normal shader ???
//  BGRA format in the mix but swapping x and z doesnt cause a match
//  CCW triangle winding maybe
//
//prd.result = make_float3(-n.z*0.5f + 0.5f,-n.y*0.5f + 0.5f, -n.x*0.5f + 0.5f ); // normal shader
//prd.result = make_float3(n.x*0.5f + 0.5f, n.y*0.5f + 0.5f, n.z*0.5f + 0.5f );   // normal shader
//prd.result = make_float3( 1.f, 0.f, 0.f ); //red
//prd.result = make_float3( 0.f, 1.f, 0.f ); //green
//prd.result = make_float3( 0.f, 0.f, 1.f ); //blue
//prd.result = make_float3(0.5f);            
//prd.result = make_float3( instanceIdentity.x/13000.f ) ;  // nodeIndex 
//prd.result = make_float3( instanceIdentity.y/250.f ) ;    // meshIndex
//prd.result = make_float3( instanceIdentity.z/50.f ) ;     // boundaryIndex
//prd.result = make_float3( instanceIdentity.w/1000.f ) ;   // sensorIndex  : need to use near clipping to see inside the PMTs to see anything
//prd.result = contrast_color ;   // according to boundary index, currently only one color as only one material ?
//prd.result = make_float3( boundaryIndex/50.f );  // grey scale according to boundary "boundary" index
//prd.result = make_float3(0.f);
// if(cos_theta > 0.0f ) prd.result.x = 0.5f ; 
//
//
// make back faces reddish : given that the "light" is effectively coming from
// the viewpoint this can probably only happen due to a geometry bug 
// Nope, no bug needed : just shooting rays from inside objects should do this.
//
// * maybe surfaces too close to each other resulting in numerical precision flipping
//   between alternate closest hit surfaces  
// * flipped triangle winding order is not impossible
// 
// Little red is seen:
//
// * small red triangles at ends of some struts/ribs on top of AD
// * when enter inside a PMT, see a concentric circle bullseye red/white pattern
//   no problem is apparent for the external view of the PMT 
// * from inside calibration assemblies quite a lot of speckly red/black
// 

#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#if OPTIX_VERSION >= 60000

using namespace optix;

// inputs from OGeo
rtBuffer<uint3>   indexBuffer; 
rtBuffer<float3> vertexBuffer;     
rtBuffer<uint4>  identityBuffer; 
rtDeclareVariable(unsigned int, instance_index,  ,);
rtDeclareVariable(unsigned int, primitive_count, ,);

// attribute variables communicating from intersection program to closest hit program
// (must be set between rtPotentialIntersection and rtReportIntersection)
//
// hmm but what about GeometryTriangles triangle_attributes ?

rtDeclareVariable(uint4, instanceIdentity,   attribute instance_identity,);
rtDeclareVariable(float3, geometricNormal, attribute geometric_normal, ); 


RT_PROGRAM void triangle_attributes()
{
    const int primIdx = rtGetPrimitiveIndex() ;
    const uint3  index  = indexBuffer[primIdx];

    const float3 p0    = vertexBuffer[index.x];
    const float3 p1    = vertexBuffer[index.y];
    const float3 p2    = vertexBuffer[index.z];
    const float3 norm    = optix::normalize(optix::cross( p1 - p0, p2 - p0 ));
    const uint4 identity = identityBuffer[instance_index*primitive_count+primIdx] ;  // index just primIdx for non-instanced

    geometricNormal = norm ;
    instanceIdentity = identity ;
}


#endif



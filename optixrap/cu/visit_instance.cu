#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, instance_bounding_radius , , );

//#define WITH_PRINT 1


RT_PROGRAM void visit_instance()
{
    const float distance = length( ray.origin ) ;  // Visit program ray.origin is in OBJECT frame
    const unsigned level = distance < instance_bounding_radius ? 0u : 1u ;  
    rtIntersectChild( level );
}

RT_PROGRAM void visit_instance_WORLD()
{
    /*
    Transform gymnastics here actually pointless... 
    No need to convert between frames, OBJECT -> WORLD 
    just directly use OBJECT frame ray.origin to yield 
    the same distance.

        0  1  2  3
        4  5  6  7
        8  9 10 11
       12 13 14 15 
    */
    
    float matrix[16] ;
    rtGetTransform( RT_OBJECT_TO_WORLD , matrix ) ;
    const float3 ipos = make_float3( matrix[3], matrix[7], matrix[11] );  // 12 13 14 was (0,0,0)
    const float3 orig = rtTransformPoint( RT_OBJECT_TO_WORLD, ray.origin );
    const float3 offset = orig - ipos  ;  
    const float distance = length( offset ) ; 
    const float distance1 = length( ray.origin ) ; 
    const unsigned level = distance < instance_bounding_radius ? 0u : 1u ;  

#ifdef WITH_PRINT
    rtPrintf("visit_instance_WORLD: level %d instance_bounding_radius %10.3f distance %10.3f distance1 %10.3f  orig (%10.3f %10.3f %10.3f) ipos (%10.3f %10.3f %10.3f)  \n", 
          level, 
          instance_bounding_radius,
          distance,
          distance1,
          orig.x, orig.y, orig.z,
          ipos.x, ipos.y, ipos.z
       ); 
    
#endif

    rtIntersectChild( level );
}




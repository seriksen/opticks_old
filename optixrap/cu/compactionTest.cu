#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include <optix_world.h>
#include "quad.h"

using namespace optix;

rtBuffer<float4>   photon_buffer ;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
//rtDeclareVariable(unsigned int,  PNUMQUAD, , );   // in OptiX 501 510 this comes out zero 
rtDeclareVariable(uint2,  compaction_param, , );

//#define WITH_PRINT 1


RT_PROGRAM void compactionTest()
{
    unsigned photon_id = launch_index.x ;  
    unsigned photon_offset = photon_id*compaction_param.x ; 

    union quad q0,q1,q2,q3 ;

    q0.f = photon_buffer[photon_offset+0] ;   
    q1.f = photon_buffer[photon_offset+1] ;   
    q2.f = photon_buffer[photon_offset+2] ;   
    q3.f = photon_buffer[photon_offset+3] ;   

#ifdef WITH_PRINT
    rtPrintf("compactionTest.cu  %5u %5u fffu(%10f, %10f, %10f, %u) \n", photon_id, photon_offset, q0.f.x, q1.f.y, q2.f.z, q3.u.w );
#endif
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}




#include "hip/hip_runtime.h"

#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

#include "boundary_lookup.h"

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtBuffer<float4,2>  out_buffer;



RT_PROGRAM void OInterpolationTest()
{
    unsigned nj = BOUNDARY_NUM_MATSUR ;
    unsigned nk = BOUNDARY_NUM_FLOAT4 ;

    uint2 out_index ; 
    int w = int(launch_index.x) ;  // 0:39 wavelength sample index
    int i = int(launch_index.y) ;  // 0:123 bnd index

    out_index.x = w ;  
    float nm = boundary_domain.x + w*boundary_domain.z ;


    for(unsigned j=0 ; j < nj ; j++)
    {
        unsigned line = i*nj + j ;  
        for(unsigned k=0 ; k < nk ; k++)
        {
           out_index.y = boundary_lookup_linek(line, k );  
           out_buffer[out_index] = boundary_lookup(nm, line, k ) ; 
        }
    }

    // NB 
    //    out_buffer.x same dim as launch.x (width), 
    //    out_buffer.y eight times larger than launch.y (height) to match the above loops

}


RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}

/*

       #float4
            |     ___ wavelength samples
            |    /
   (123, 4, 2, 39, 4)
    |    |          \___ float4 props        
  #bnd   | 
         |
    omat/osur/isur/imat  
*/



#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"  // make_color
#include "color_lookup.h"
#include "hemi-pmt.h"
#include "PerRayData_radiance.h"

using namespace optix;

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        front, , );

rtDeclareVariable(float4,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned int,  parallel, , );

rtBuffer<uchar4, 2>              output_buffer;
//rtBuffer<float, 2>               depth_buffer;


rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  resolution_scale, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;


rtDeclareVariable(unsigned int,  touch_mode, , );
rtDeclareVariable(unsigned int,  touch_bad, , );
rtDeclareVariable(uint2,         touch_index,  , );
rtDeclareVariable(uint2,         touch_dim,  , );
rtBuffer<uint4,2>         touch_buffer;

// BGRA
#define BLUE  make_uchar4(255u,  0u,  0u,255u)
#define GREEN make_uchar4(  0u,255u,  0u,255u)
#define RED   make_uchar4(  0u,  0u,255u,255u)


RT_PROGRAM void pinhole_camera()
{

  PerRayData_radiance prd;
  prd.flag = 0u ; 
  prd.result = bad_color ;

  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f ;

  optix::Ray ray = parallel == 0 ? 
                       optix::make_Ray( eye                 , normalize(d.x*U + d.y*V + W), radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX)
                     :
                       optix::make_Ray( eye + d.x*U + d.y*V , normalize(W)                , radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX)
                     ;


  // (d.x,d.y) spans screen pixels (-1:1,-1:1) 
  //
  // oglrap-/Composition::getEyeUVW provides : eye,U,V,W
  //
  //   * eye : world frame position of camera  
  //   * U,V : vectors defining dimension and direction of x,y 
  //           axes of image plane expresses in world frame
  //   * W   : is eye to image plane direction and dimension   
  // 
  // scene_epsilon is "t_min" but ray_direction is normalized, 
  // so that makes "t_min" a world frame quantity, which makes 
  // setting it to world frame camera Near to be appropriate 
  // (when looking straight ahead at least). 
  //
  // They are not really equivalent though, near being the distance
  // to the screen parallel frustum plane whereas 
  // scene_epsilon is the distance along the ray at which to start 
  // accepting intersections ?
  //


#if RAYTRACE_TIMEVIEW
  clock_t t0 = clock(); 
#endif

  rtTrace(top_object, ray, prd);

#if RAYTRACE_TIMEVIEW
  clock_t t1 = clock(); 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  uchar4  color = = make_color( make_float3(  pixel_time ) ); 
#else
  uchar4 color = make_color( prd.result ) ; // BGRA
#endif

  if( resolution_scale == 1)  
  { 
      output_buffer[launch_index] = color ; 
     // depth_buffer[launch_index] = zHit_clip ; 
  }
  else if( resolution_scale == 2)
  {
      unsigned int wx2 = 2*launch_index.x ; 
      unsigned int wy2 = 2*launch_index.y ; 

      uint2 idx00 = make_uint2(wx2  , wy2) ; 
      uint2 idx10 = make_uint2(wx2+1, wy2) ; 
      uint2 idx01 = make_uint2(wx2  , wy2+1) ; 
      uint2 idx11 = make_uint2(wx2+1, wy2+1) ; 

      output_buffer[idx00] = color ; 
      output_buffer[idx10] = color ; 
      output_buffer[idx01] = color ; 
      output_buffer[idx11] = color ; 

      //depth_buffer[idx00] = zHit_clip ; 
      //depth_buffer[idx10] = zHit_clip ; 
      //depth_buffer[idx01] = zHit_clip ; 
      //depth_buffer[idx11] = zHit_clip ; 

  }
  else if( resolution_scale > 2)
  {
      unsigned int wx = resolution_scale*launch_index.x ; 
      unsigned int wy = resolution_scale*launch_index.y ; 
      for(unsigned int i=0 ; i < resolution_scale ; i++){
      for(unsigned int j=0 ; j < resolution_scale ; j++){
          uint2 idx = make_uint2(wx+i, wy+j) ; 
          output_buffer[idx] = color ; 
          //depth_buffer[idx] = zHit_clip ; 
      }
      }
  }
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  //rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}




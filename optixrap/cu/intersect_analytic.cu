#include "hip/hip_runtime.h"
#include "OpticksCSG.h"
#include "NPart.h"

#include <optix_world.h>

#include "quad.h"
#include "Part.h"

#include "switches.h"
#define DEBUG 1

#include "boolean_solid.h"
#include "hemi-pmt.h"

// CUDART_ defines
#include "hip/hip_math_constants.h"

using namespace optix;

// generated from /Users/blyth/opticks/optixrap/cu by boolean_h.py on Sat Mar  4 20:37:03 2017 
rtDeclareVariable(uint4, packed_boolean_lut_ACloser, , ) = { 0x22121141, 0x00014014, 0x00141141, 0x00000000 } ; 
rtDeclareVariable(uint4, packed_boolean_lut_BCloser, , ) = { 0x22115122, 0x00022055, 0x00133155, 0x00000000 } ; 



static __device__
int boolean_ctrl_packed_lookup( OpticksCSG_t operation, IntersectionState_t stateA, IntersectionState_t stateB, bool ACloser )
{
    const uint4& lut = ACloser ? packed_boolean_lut_ACloser : packed_boolean_lut_BCloser ;
    unsigned offset = 3*(unsigned)stateA + (unsigned)stateB ;   
    unsigned index = (unsigned)operation - (unsigned)CSG_UNION ; 
    return offset < 8 ? (( getByIndex(lut, index) >> (offset*4)) & 0xf) : CTRL_RETURN_MISS ;
}


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, t_parameter, rtIntersectionDistance, );
rtDeclareVariable(float, propagate_epsilon, , );


rtDeclareVariable(unsigned int, instance_index,  ,);
// optix::GeometryInstance instance_index into the identity buffer, 
// set by oxrap/OGeo.cc, 0 for non-instanced 

rtDeclareVariable(unsigned int, primitive_count, ,);
// TODO: instanced analytic identity, using the above and below solid level identity buffer

rtBuffer<Part> partBuffer; 
rtBuffer<Matrix4x4> tranBuffer; 

rtBuffer<uint4>  primBuffer; 
rtBuffer<uint4>  identityBuffer;   // from GMergedMesh::getAnalyticInstanceIdentityBuffer()
rtBuffer<float4> prismBuffer ;


// attributes communicate to closest hit program,
// they must be set inbetween rtPotentialIntersection and rtReportIntersection

rtDeclareVariable(uint4, instanceIdentity,   attribute instance_identity,);
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 



#include "bbox.h"
#include "transform_test.h"

#include "csg_intersect_primitive.h"
#include "csg_intersect_part.h"
#include "csg_intersect_boolean.h"

#include "intersect_ztubs.h"
#include "intersect_zsphere.h"
#include "intersect_box.h"
#include "intersect_prism.h"


/*
TODO
~~~~~~

* use prim.z for numTran, instead of duplicating primIdx 

*/

RT_PROGRAM void bounds (int primIdx, float result[6])
{
    if(primIdx == 0) 
    { 
        transform_test();
    }

    unsigned tranBuffer_size = tranBuffer.size() ;
    const uint4& prim    = primBuffer[primIdx]; 

    unsigned partOffset  = prim.x ;  
    unsigned numParts    = prim.y ; 
    unsigned primFlag    = prim.w ;  

    unsigned height = TREE_HEIGHT(numParts) ; // 1->0, 3->1, 7->2, 15->3, 31->4 
    unsigned numNodes = TREE_NODES(height) ;      

    rtPrintf("##bounds primIdx %2d partOffset %2d numParts %2d height %2d numNodes %2d tranBuffer_size %3u \n", primIdx, partOffset, numParts, height, numNodes, tranBuffer_size );

    uint4 identity = identityBuffer[instance_index] ;  // instance_index from OGeo is 0 for non-instanced

    optix::Aabb* aabb = (optix::Aabb*)result;
    *aabb = optix::Aabb();

    if(primFlag == CSG_FLAGNODETREE)  
    {
        unsigned nodeIdx = 1 << height ; 
        while(nodeIdx)
        {
            int depth = TREE_DEPTH(nodeIdx) ;
            int elev = height - depth ; 

            Part pt = partBuffer[partOffset+nodeIdx-1];  // nodeIdx is 1-based

            unsigned typecode = pt.typecode() ; 
            unsigned gtransformIdx = pt.gtransformIdx() ;  //  gtransformIdx is 1-based, 0 meaning None
    
            rtPrintf("## bounds nodeIdx %2u depth %2d elev %2d typecode %2u gtransformIdx %2u \n", nodeIdx, depth, elev, typecode, gtransformIdx );

            if(gtransformIdx == 0)
            {
                switch(typecode)
                {
                    case CSG_SPHERE: csg_bounds_sphere(pt.q0, aabb, NULL  );  break ;
                    case CSG_BOX:    csg_bounds_box(pt.q0, aabb, NULL  );     break ;
                    case CSG_SLAB:   csg_bounds_slab(  pt.q0, pt.q1, aabb, NULL ) ; break ;  /* infinite slabs must always be used in intersection */
                    case CSG_PLANE:  csg_bounds_plane(  pt.q0, aabb, NULL ) ; break ;       /* infinite plane must always be used in intersection */
                    default:                                                  break ; 
                }
            }
            else
            {
                unsigned trIdx = 3*(gtransformIdx-1)+0 ;
                if(trIdx >= tranBuffer_size)
                { 
                    rtPrintf("## bounds ABORT trIdx %3u overflows tranBuffer_size %3u \n", trIdx, tranBuffer_size );
                    return ;  
                }
                optix::Matrix4x4 tr = tranBuffer[trIdx] ; 
                switch(typecode)
                {
                    case CSG_SPHERE: csg_bounds_sphere(pt.q0, aabb, &tr  );  break ;
                    case CSG_BOX:    csg_bounds_box(   pt.q0, aabb, &tr  );  break ;
                    case CSG_SLAB:   csg_bounds_slab(  pt.q0, pt.q1, aabb, &tr ) ; break ;  /* infinite slabs must always be used in intersection */
                    case CSG_PLANE:  csg_bounds_plane( pt.q0, aabb, &tr )   ; break ;       /* infinite plane must always be used in intersection */
                    default:                                                 break ; 
                }
            }

            nodeIdx = POSTORDER_NEXT( nodeIdx, elev ) ;
            // see opticks/dev/csg/postorder.py for explanation of bit-twiddling postorder  
            //unsigned nodeIdx2 = nodeIdx & 1 ? nodeIdx >> 1 : (nodeIdx << elev) + (1 << elev) ;
            //if(nodeIdx2 != nodeIdx) rtPrintf("nodeIdx MISMATCH \n");
        }
    }
    else if(primFlag == CSG_FLAGPARTLIST)  
    {
        for(unsigned int p=0 ; p < numParts ; p++)
        { 
            Part pt = partBuffer[partOffset + p] ; 
            unsigned typecode = pt.typecode() ; 

            identity.z = pt.boundary() ;  // boundary from partBuffer (see ggeo-/GPmt)

            if(typecode == CSG_PRISM) 
            {
                make_prism(pt.q0.f, aabb) ;
            }
            else
            {
                aabb->include( make_float3(pt.q2.f), make_float3(pt.q3.f) );
            }
        } 
    }
    else
    {
        rtPrintf("## intersect_analytic.cu:bounds ABORT BAD primflag %d \n", primFlag );
        return ; 
    }
    rtPrintf("##intersect_analytic.cu:bounds primIdx %d primFlag %d min %10.4f %10.4f %10.4f max %10.4f %10.4f %10.4f \n", primIdx, primFlag, 
        result[0],
        result[1],
        result[2],
        result[3],
        result[4],
        result[5]
        );

}


/**

identityBuffer
~~~~~~~~~~~~~~~~

* just placeholder zeros for analytic test geometry 

* setting identity.z adopts boundary index from partBuffer, see npy/NPart.hpp for layout (also GPmt)
  at intersections the uint4 identity is copied into the instanceIdentity attribute,
  hence making it available to material1_propagate.cu:closest_hit_propagate
  where crucially the instanceIdentity.z -> boundaryIndex


**/


RT_PROGRAM void intersect(int primIdx)
{
    const uint4& prim    = primBuffer[primIdx]; 

    unsigned partOffset  = prim.x ;  
    unsigned numParts    = prim.y ; 
    unsigned primFlag    = prim.w ;  

    uint4 identity = identityBuffer[instance_index] ; 


    if(primFlag == CSG_FLAGNODETREE)  
    { 
        Part pt = partBuffer[partOffset] ; 

        identity.z = pt.boundary() ;        // replace placeholder zero with test analytic geometry root node boundary

        evaluative_csg( prim, identity );
        //intersect_csg( prim, identity );

    }
    else if(primFlag == CSG_FLAGPARTLIST)  
    {
        for(unsigned int p=0 ; p < numParts ; p++)
        {  
            Part pt = partBuffer[partOffset + p] ; 

            identity.z = pt.boundary() ;   

            unsigned typecode = pt.typecode() ; 

            switch(typecode)
            {
                case CSG_ZERO:
                    intersect_aabb(pt.q2, pt.q3, identity);
                    break ; 
                case CSG_SPHERE:
                    intersect_zsphere<false>(pt.q0,pt.q1,pt.q2,pt.q3,identity);
                    break ; 
                case CSG_TUBS:
                    intersect_ztubs(pt.q0,pt.q1,pt.q2,pt.q3,identity);
                    break ; 
                case CSG_BOX:
                    intersect_box(pt.q0,identity);
                    break ; 
                case CSG_PRISM:
                    // q0.f param used in *bounds* to construct prismBuffer, which is used within intersect_prism
                    intersect_prism(identity);
                    break ; 
            }
        }
    } 
}



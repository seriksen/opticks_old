#include "hip/hip_runtime.h"
#include "OpticksCSG.h"
#include "NPart.h"

#include <optix_world.h>

#include "quad.h"
#include "Part.h"

#include "switches.h"
#define DEBUG 1

#include "boolean_solid.h"
#include "hemi-pmt.h"

// CUDART_ defines
#include "hip/hip_math_constants.h"

using namespace optix;

// generated from /Users/blyth/opticks/optixrap/cu by boolean_h.py on Sat Mar  4 20:37:03 2017 
rtDeclareVariable(uint4, packed_boolean_lut_ACloser, , ) = { 0x22121141, 0x00014014, 0x00141141, 0x00000000 } ; 
rtDeclareVariable(uint4, packed_boolean_lut_BCloser, , ) = { 0x22115122, 0x00022055, 0x00133155, 0x00000000 } ; 



static __device__
int boolean_ctrl_packed_lookup( OpticksCSG_t operation, IntersectionState_t stateA, IntersectionState_t stateB, bool ACloser )
{
    const uint4& lut = ACloser ? packed_boolean_lut_ACloser : packed_boolean_lut_BCloser ;
    unsigned offset = 3*(unsigned)stateA + (unsigned)stateB ;   
    unsigned index = (unsigned)operation - (unsigned)CSG_UNION ; 
    return offset < 8 ? (( getByIndex(lut, index) >> (offset*4)) & 0xf) : CTRL_RETURN_MISS ;
}


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, t_parameter, rtIntersectionDistance, );
rtDeclareVariable(float, propagate_epsilon, , );


rtDeclareVariable(unsigned int, instance_index,  ,);
// optix::GeometryInstance instance_index into the identity buffer, 
// set by oxrap/OGeo.cc, 0 for non-instanced 

rtDeclareVariable(unsigned int, primitive_count, ,);
// TODO: instanced analytic identity, using the above and below solid level identity buffer

rtBuffer<Part> partBuffer; 
rtBuffer<Matrix4x4> tranBuffer; 

rtBuffer<uint4>  primBuffer; 
rtBuffer<uint4>  identityBuffer;   // from GMergedMesh::getAnalyticInstanceIdentityBuffer()
rtBuffer<float4> prismBuffer ;


// attributes communicate to closest hit program,
// they must be set inbetween rtPotentialIntersection and rtReportIntersection

rtDeclareVariable(uint4, instanceIdentity,   attribute instance_identity,);
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 



#include "bbox.h"
#include "transform_test.h"

#include "csg_intersect_part.h"
#include "csg_intersect_boolean.h"

#include "intersect_ztubs.h"
#include "intersect_zsphere.h"
#include "intersect_box.h"
#include "intersect_prism.h"


/*
TODO
~~~~~~

* use prim.z for numTran, instead of duplicating primIdx 

*/

RT_PROGRAM void bounds (int primIdx, float result[6])
{
    if(primIdx == 0) 
    { 
        transform_test();
    }

    unsigned tranBuffer_size = tranBuffer.size() ;
    const uint4& prim    = primBuffer[primIdx]; 

    unsigned partOffset  = prim.x ;  
    unsigned numParts    = prim.y ; 
    unsigned primFlag    = prim.w ;  

    unsigned height = TREE_HEIGHT(numParts) ; // 1->0, 3->1, 7->2, 15->3, 31->4 
    unsigned numNodes = TREE_NODES(height) ;      

    rtPrintf("##bounds primIdx %2d partOffset %2d numParts %2d height %2d numNodes %2d tranBuffer_size %3u \n", primIdx, partOffset, numParts, height, numNodes, tranBuffer_size );

    uint4 identity = identityBuffer[instance_index] ;  // instance_index from OGeo is 0 for non-instanced

    optix::Aabb* aabb = (optix::Aabb*)result;
    *aabb = optix::Aabb();

    if(primFlag == CSG_FLAGNODETREE)  
    {
        unsigned nodeIdx = 1 << height ; 
        while(nodeIdx)
        {
            int depth = TREE_DEPTH(nodeIdx) ;
            int elev = height - depth ; 

            Part pt = partBuffer[partOffset+nodeIdx-1];  // nodeIdx is 1-based

            unsigned partType = pt.partType() ; 
            unsigned gtransformIdx = pt.gtransformIdx() ;  //  gtransformIdx is 1-based, 0 meaning None
    
            rtPrintf("## bounds nodeIdx %2u depth %2d elev %2d partType %2u gtransformIdx %2u \n", nodeIdx, depth, elev, partType, gtransformIdx );

            if(gtransformIdx == 0)
            {
                switch(partType)
                {
                    case CSG_SPHERE: csg_bounds_sphere(pt.q0, aabb, NULL  );  break ;
                    case CSG_BOX:    csg_bounds_box(pt.q0, aabb, NULL  );     break ;
                    default:                                                  break ; 
                }
            }
            else
            {
                unsigned trIdx = 2*(gtransformIdx-1)  ; 
                if(trIdx >= tranBuffer_size)
                { 
                    rtPrintf("## bounds ABORT trIdx %3u overflows tranBuffer_size %3u \n", trIdx, tranBuffer_size );
                    return ;  
                }
                optix::Matrix4x4 tr = tranBuffer[trIdx] ; 
                switch(partType)
                {
                    case CSG_SPHERE: csg_bounds_sphere(pt.q0, aabb, &tr  );  break ;
                    case CSG_BOX:    csg_bounds_box(   pt.q0, aabb, &tr  );  break ;
                    default:                                                 break ; 
                }
            }

            nodeIdx = nodeIdx & 1 ? nodeIdx >> 1 : (nodeIdx << elev) + (1 << elev) ;
            // see opticks/dev/csg/postorder.py for explanation of bit-twiddling postorder  
        }
    }
    else if(primFlag == CSG_FLAGPARTLIST)  
    {
        for(unsigned int p=0 ; p < numParts ; p++)
        { 
            Part pt = partBuffer[partOffset + p] ; 
            unsigned partType = pt.q2.u.w ; 

            identity.z = pt.q1.u.z ;  // boundary from partBuffer (see ggeo-/GPmt)

            if(partType == CSG_PRISM) 
            {
                make_prism(pt.q0.f, aabb) ;
            }
            else
            {
                aabb->include( make_float3(pt.q2.f), make_float3(pt.q3.f) );
            }
        } 
    }
    else
    {
        rtPrintf("## intersect_analytic.cu:bounds ABORT BAD primflag %d \n", primFlag );
        return ; 
    }
    rtPrintf("##intersect_analytic.cu:bounds primIdx %d primFlag %d min %10.4f %10.4f %10.4f max %10.4f %10.4f %10.4f \n", primIdx, primFlag, 
        result[0],
        result[1],
        result[2],
        result[3],
        result[4],
        result[5]
        );

}


/**

identityBuffer
~~~~~~~~~~~~~~~~

* just placeholder zeros for analytic test geometry 

* setting identity.z adopts boundary index from partBuffer, see npy/NPart.hpp for layout (also GPmt)
  at intersections the uint4 identity is copied into the instanceIdentity attribute,
  hence making it available to material1_propagate.cu:closest_hit_propagate
  where crucially the instanceIdentity.z -> boundaryIndex


**/


RT_PROGRAM void intersect(int primIdx)
{
    const uint4& prim    = primBuffer[primIdx]; 

    unsigned partOffset  = prim.x ;  
    unsigned numParts    = prim.y ; 
    unsigned primFlag    = prim.w ;  

    uint4 identity = identityBuffer[instance_index] ; 


    if(primFlag == CSG_FLAGNODETREE)  
    { 
        Part pt = partBuffer[partOffset] ; 

        identity.z = pt.q1.u.z ;        // replace placeholder zero with test analytic geometry root node boundary

        evaluative_csg( prim, identity );
        //intersect_csg( prim, identity );

    }
    else if(primFlag == CSG_FLAGPARTLIST)  
    {
        for(unsigned int p=0 ; p < numParts ; p++)
        {  
            Part pt = partBuffer[partOffset + p] ; 

            identity.z = pt.q1.u.z ;   

            unsigned partType = pt.q2.u.w ; 

            switch(partType)
            {
                case CSG_ZERO:
                    intersect_aabb(pt.q2, pt.q3, identity);
                    break ; 
                case CSG_SPHERE:
                    intersect_zsphere<false>(pt.q0,pt.q1,pt.q2,pt.q3,identity);
                    break ; 
                case CSG_TUBS:
                    intersect_ztubs(pt.q0,pt.q1,pt.q2,pt.q3,identity);
                    break ; 
                case CSG_BOX:
                    intersect_box(pt.q0,identity);
                    break ; 
                case CSG_PRISM:
                    // q0.f param used in *bounds* to construct prismBuffer, which is used within intersect_prism
                    intersect_prism(identity);
                    break ; 
            }
        }
    } 
}



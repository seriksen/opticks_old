#include "hip/hip_runtime.h"
#include "OpticksCSG.h"

// shape flag enums from npy-
#include "NPart.h"
#include "NCylinder.h"
#include "NSlab.h"
#include "NZSphere.h"

#include <optix_world.h>

#include "quad.h"
#include "Part.h"
#include "Prim.h"

#include "switches.h"
#define DEBUG 1

#include "boolean_solid.h"
#include "hemi-pmt.h"

// CUDART_ defines
#include "hip/hip_math_constants.h"

using namespace optix;

rtBuffer<Matrix4x4> tranBuffer; 

#include "bbox.h"

//#define CSG_INTERSECT_CONE_TEST 1
//#define CSG_INTERSECT_CONVEXPOLYHEDRON_TEST 1
#define CSG_INTERSECT_TORUS_TEST 1
#include "csg_intersect_primitive.h"

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
//rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<float4>  output_buffer;

RT_PROGRAM void intersect_analytic_test()
{
    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 

    rtPrintf("## intersect_analytic_test %llu\n", photon_id);

    //csg_intersect_cone_test(photon_id);
    //csg_intersect_convexpolyhedron_test(photon_id);
    csg_intersect_torus_test_0(photon_id);
    //csg_intersect_sphere_test(photon_id);
    
    output_buffer[photon_offset+0] = make_float4(40.f, 40.f, 40.f, 40.f);
    output_buffer[photon_offset+1] = make_float4(41.f, 41.f, 41.f, 41.f);
    output_buffer[photon_offset+2] = make_float4(42.f, 42.f, 42.f, 42.f);
    output_buffer[photon_offset+3] = make_float4(43.f, 43.f, 43.f, 43.f);

}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();

    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 
    
    output_buffer[photon_offset+0] = make_float4(-40.f, -40.f, -40.f, -40.f);
    output_buffer[photon_offset+1] = make_float4(-41.f, -41.f, -41.f, -41.f);
    output_buffer[photon_offset+2] = make_float4(-42.f, -42.f, -42.f, -42.f);
    output_buffer[photon_offset+3] = make_float4(-43.f, -43.f, -43.f, -43.f);

}




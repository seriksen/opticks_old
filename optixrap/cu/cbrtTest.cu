#include "hip/hip_runtime.h"
/*

For optixtest bash function to build and run see ../tests/OptixMinimalTest.hh 

This succeeds to reproduce the segv within createProgramFromPTXFile
within intersect_analytic_test arising from SolveCubicNumericalRecipe.h cbrt(double). 
But to do so had to almost duplicate it entirely.

* current working assumption is that double precision trig/cbrt etc functions
  are heavy on the GPU requiring a lot of code, hence there is a tendency for
  some particulary heavy functions like cbrt(double) to be the last straw
  that breaks the camels back

  * evidence for this came from cbrtTest.cu where repeating a function
    call triggered the segv

* a promising solution to this issue is to reuse heavy double precision  
  math functions like SolveCubic into RT_CALLABLE_PROGRAM to avoid the duplication

* https://devtalk.nvidia.com/default/topic/735307/optix/strange-error-while-reading-a-ptx-file/2
* https://devtalk.nvidia.com/default/topic/764148/double-precision-trigonometric-functions/#4282733
* https://devtalk.nvidia.com/search/more/sitecommentsearch/optix%20double%20precision/

*/

#include <optix_world.h>

rtBuffer<rtCallableProgramId<unsigned(double,double,double,double*,unsigned)> > callable ;


#define SOLVE_QUARTIC_DEBUG 1
typedef double Solve_t ; 
#include "SolveCubicNumericalRecipe.h"

/*
#define SOLVE_QUARTIC_DEBUG 1
typedef double Solve_t ; 
#include "Solve.h"
*/


/*
#include "quad.h"
#include "bbox.h"
#define CSG_INTERSECT_TORUS_TEST 1
#include "csg_intersect_torus.h"
*/


using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4>  output_buffer;


RT_PROGRAM void cbrtTest()
{
    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 
   

    unsigned msk = 0u ; 
    double a = 10.0 ; 
    double b = 20.0 ; 
    double c = 30.0 ; 

    double rts[3] ; 
    unsigned nrts ; 

    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 
    nrts = callable[0](a, b, c, rts, msk); 

#ifdef CSG_INTERSECT_TORUS_TEST 
    rtPrintf("cbrtTest SolveCubic_callable a:%f b:%f c:%f nrts:%u rts (%g %g %g)  \n", a,b,c,nrts,rts[0],rts[1],rts[2] );
#endif
   

    float twentyseven_f = 27.f ; 
    float crf = cbrtf(twentyseven_f);

    double twentyseven_d(27) ; 
    double crd = cbrt(twentyseven_d);

    Solve_t p,q,r ;   

    p = 49526.79994 ;        
    q = 408572956.1 ;
    r = -1483476.478 ;

    Solve_t roq = -r/q ; 

#ifdef CSG_INTERSECT_TORUS_TEST 
    rtPrintf("cbrtTest crf:%f crd:%g  \n", crf, crd );
    rtPrintf("SolveCubicTest pqr (%15g %15g %15g)  x^3 + p x^2 + q x + r = 0   -r/q %g   \n", p,q,r, roq );
#endif
 

    unsigned nr = 0 ;  
    Solve_t xx[3] ; 
    nr = SolveCubic(p,q,r,xx, 0u ); 
    nr = SolveCubic(p,q,r,xx, 0u ); 
    //nr = SolveCubic(p,q,r,xx, 0u ); 

    // NB : only one or 2 calls to simple inlined SolveCubic works 
    //      before getting segv in createProgramFromPTX 
    //      
    //      contrast with above callable approach where calling many times seems
    //      is not much of a resource burden
    //      
    //      It appears the heavy nature of GPU double precision math can be tamed 
    //      by doing it via callables. 
    //
    // HMM : doing twice works with default stacksize of 1024, more than twice segv in createProgramFromPTX
    //       three times works with 2* stacksize 


#ifdef CSG_INTERSECT_TORUS_TEST 
    rtPrintf("nr %u  \n", nr ) ;

    for(unsigned i=0 ; i < nr ; i++)
    {
        Solve_t x = xx[i] ; 

        Solve_t x3 = x*x*x ; 
        Solve_t x2 = p*x*x ; 
        Solve_t x1 = q*x ; 
        Solve_t x0 = r ; 

        Solve_t x3_x2 = x3 + x2 ; 
        Solve_t x1_x0 = x1 + x0 ;
        Solve_t x3_x2_x1_x0 = x3_x2 + x1_x0 ;
  
        Solve_t residual = ((x + p)*x + q)*x + r ; 
        rtPrintf("xx[%u] = %15g  residual %15g  x3210 (%15g %15g %15g %15g) x3_x2 %15g x1_x0 %15g x3_x2_x1_x0 %15g    \n", i, xx[i], residual, x3, x2, x1, x0, x3_x2, x1_x0, x3_x2_x1_x0 ) ;
    }
#endif


    // WOW: calling "csg_intersect_torus_scale_test" twice is prone to: Segmentation fault: 11   within createPTXFromFile
    //csg_intersect_torus_scale_test(photon_id, false);
    //csg_intersect_torus_scale_test(photon_id, true);


    output_buffer[photon_offset+0] = make_float4(40.f, 40.f, 40.f, 40.f);
    output_buffer[photon_offset+1] = make_float4(41.f, 41.f, 41.f, 41.f);
    output_buffer[photon_offset+2] = make_float4(42.f, 42.f, 42.f, 42.f);
    output_buffer[photon_offset+3] = make_float4(43.f, 43.f, 43.f, 43.f);
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();

    unsigned long long photon_id = launch_index.x ;  
    unsigned int photon_offset = photon_id*4 ; 
    
    output_buffer[photon_offset+0] = make_float4(-40.f, -40.f, -40.f, -40.f);
    output_buffer[photon_offset+1] = make_float4(-41.f, -41.f, -41.f, -41.f);
    output_buffer[photon_offset+2] = make_float4(-42.f, -42.f, -42.f, -42.f);
    output_buffer[photon_offset+3] = make_float4(-43.f, -43.f, -43.f, -43.f);
}


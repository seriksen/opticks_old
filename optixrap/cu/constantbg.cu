#include "hip/hip_runtime.h"
#include <optix_world.h>

rtDeclareVariable(float4, bg_color, , );

struct PerRayData_radiance
{
  float4 result;
  float importance;
  int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}

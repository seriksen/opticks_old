#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include <optix_world.h>

rtDeclareVariable(float4, bg_color, , );

struct PerRayData_radiance
{
  float4 result;
  float importance;
  int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}

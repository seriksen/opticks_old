#include "OpIndexer.hh"

#include <cassert>

// optickscore-
#include "OpticksConst.hh"  
#include "OpticksEvent.hh"  

// npy-
#include "Timer.hpp"  
#include "NPY.hpp"  

// optixrap-
#include "OBuf.hh"

// cudawrap-
#include "CResource.hh"
#include "CBufSpec.hh"
#include "CBufSlice.hh"

// thrustrap-
#include "TBuf.hh"
#include "TSparse.hh"
#include "TUtil.hh"

// thrust 
#include <thrust/device_vector.h>


#define TIMER(s) \
    { \
       if(m_evt)\
       {\
          Timer& t = *(m_evt->getTimer()) ;\
          t((s)) ;\
       }\
    }


void OpIndexer::indexSequenceViaThrust(
   TSparse<unsigned long long>& seqhis, 
   TSparse<unsigned long long>& seqmat, 
   bool verbose
)
{
    NPY<unsigned char>* phosel = m_evt->getPhoselData() ;
    NPY<unsigned char>* recsel = m_evt->getRecselData() ;

    // allocate phosel and recsel GPU buffers
    thrust::device_vector<unsigned char> dps(phosel->getNumValues());
    thrust::device_vector<unsigned char> drs(recsel->getNumValues());

    // CUDA refs to the buffers obtained from Thrust 
    //     thrustrap-/TUtil.make_bufspec 
    //     does thrust::raw_pointer_cast to get raw CUDA pointer from Thrust 
    CBufSpec rps = make_bufspec<unsigned char>(dps); 
    CBufSpec rrs = make_bufspec<unsigned char>(drs) ;

    indexSequenceImp(seqhis, seqmat, rps, rrs, verbose);
}

void OpIndexer::indexSequenceViaOpenGL(
   TSparse<unsigned long long>& seqhis, 
   TSparse<unsigned long long>& seqmat, 
   bool verbose
)
{
    NPY<unsigned char>* phosel = m_evt->getPhoselData() ;
    NPY<unsigned char>* recsel = m_evt->getRecselData() ;

    unsigned int phosel_id = phosel->getBufferId() ;
    unsigned int recsel_id = recsel->getBufferId() ;
    printf("OpIndexer::indexSequenceViaOpenGL phosel_id %u recsel_id %u \n", phosel_id, recsel_id ); 

    CResource rphosel( phosel_id, CResource::W );
    CResource rrecsel( recsel_id, CResource::W );

    // grab refs to the OpenGL GPU buffers
    CBufSpec rps = rphosel.mapGLToCUDA<unsigned char>() ;
    CBufSpec rrs = rrecsel.mapGLToCUDA<unsigned char>() ;
   
    indexSequenceImp(seqhis, seqmat, rps, rrs, verbose);

    // hand back to OpenGL
    rphosel.unmapGLToCUDA(); 
    rrecsel.unmapGLToCUDA(); 
}





void OpIndexer::indexSequenceImp(
   TSparse<unsigned long long>& seqhis, 
   TSparse<unsigned long long>& seqmat, 
   const CBufSpec& rps,
   const CBufSpec& rrs,
   bool verbose 
)
{
    // NB the make_lookup writes into constant GPU memory 
    //    so must apply that lookup before doing another 
    //    because of this cannot move the make_lookup prior
    //    to this 

    TBuf tphosel("tphosel", rps );
    tphosel.zero();

    TBuf trecsel("trecsel", rrs );

    if(verbose) dump(tphosel, trecsel);

    // phosel buffer is shaped (num_photons, 1, 4)
    CBufSlice tp_his = tphosel.slice(4,0) ; // stride, begin  
    CBufSlice tp_mat = tphosel.slice(4,1) ; 
 

    TIMER("_seqhisMakeLookup");
    seqhis.make_lookup(); 
    TIMER("seqhisMakeLookup");
    seqhis.apply_lookup<unsigned char>(tp_his); 
    TIMER("seqhisApplyLookup");

    if(verbose) dumpHis(tphosel, seqhis) ;

    TIMER("_seqmatMakeLookup");
    seqmat.make_lookup();
    TIMER("seqmatMakeLookup");
    seqmat.apply_lookup<unsigned char>(tp_mat);
    TIMER("seqmatApplyLookup");

    if(verbose) dumpMat(tphosel, seqmat) ;

    tphosel.repeat_to<unsigned char>( &trecsel, 4, 0, tphosel.getSize(), m_maxrec );  // other, stride, begin, end, repeats



    NPY<unsigned char>* phosel = m_evt->getPhoselData() ;
    NPY<unsigned char>* recsel = m_evt->getRecselData() ;

    // hmm: this pull back to host might not be necessary : only used on GPU ?
    tphosel.download<unsigned char>( phosel );  // hipMemcpyDeviceToHost
    trecsel.download<unsigned char>( recsel );
}







void OpIndexer::indexBoundariesFromOptiX(OBuf* pho, unsigned int stride, unsigned int begin)
{
     CBufSlice cbnd = pho->slice(stride,begin) ;    // gets CUDA devPtr from OptiX

     TSparse<int> boundaries(OpticksConst::BNDIDX_NAME_, cbnd, false); // hexkey effects Index and dumping only 
    
     m_evt->setBoundaryIndex(boundaries.getIndex());
    
     boundaries.make_lookup();

     if(m_verbose)
        boundaries.dump("OpIndexer::indexBoundariesFromOptiX PTR_FROM_OPTIX TSparse<int>::dump");
}

void OpIndexer::indexBoundariesFromOpenGL(unsigned int photon_id, unsigned int stride, unsigned int begin)
{
    // NB this is not using the OptiX buffer, 
    //    OpenGL buffer is interop to CUDA accessed directly 

    CResource rphoton( photon_id, CResource::R );

    CBufSpec rph = rphoton.mapGLToCUDA<int>();    // gets CUDA devPtr from OpenGL
    {
        CBufSlice cbnd = rph.slice(stride,begin) ; // stride, begin  

        TSparse<int> boundaries(OpticksConst::BNDIDX_NAME_, cbnd, false);
    
        m_evt->setBoundaryIndex(boundaries.getIndex());
    
        boundaries.make_lookup();

        if(m_verbose)
           boundaries.dump("OpIndexer::indexBoundariesFromOpenGL PTR_FROM_OPTIX TSparse<int>::dump");

        rphoton.unmapGLToCUDA(); 
    }
}




